#include "hip/hip_runtime.h"
#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/types.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace blend
    {
        __global__ void addSrcWeightKernel16S(const PtrStep<short> src, const PtrStep<short> src_weight,
            PtrStep<short> dst, PtrStep<short> dst_weight, int rows, int cols)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < rows && x < cols)
            {
                const short3 v = ((const short3*)src.ptr(y))[x];
                short w = src_weight.ptr(y)[x];
                ((short3*)dst.ptr(y))[x].x += short((v.x * w) >> 8);
                ((short3*)dst.ptr(y))[x].y += short((v.y * w) >> 8);
                ((short3*)dst.ptr(y))[x].z += short((v.z * w) >> 8);
                dst_weight.ptr(y)[x] += w;
            }
        }

        void addSrcWeightGpu16S(const PtrStep<short> src, const PtrStep<short> src_weight,
            PtrStep<short> dst, PtrStep<short> dst_weight, cv::Rect &rc)
        {
            dim3 threads(16, 16);
            dim3 grid(divUp(rc.width, threads.x), divUp(rc.height, threads.y));
            addSrcWeightKernel16S<<<grid, threads>>>(src, src_weight, dst, dst_weight, rc.height, rc.width);
            cudaSafeCall(hipGetLastError());
        }

        __global__ void addSrcWeightKernel32F(const PtrStep<short> src, const PtrStepf src_weight,
            PtrStep<short> dst, PtrStepf dst_weight, int rows, int cols)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < rows && x < cols)
            {
                const short3 v = ((const short3*)src.ptr(y))[x];
                float w = src_weight.ptr(y)[x];
                ((short3*)dst.ptr(y))[x].x += static_cast<short>(v.x * w);
                ((short3*)dst.ptr(y))[x].y += static_cast<short>(v.y * w);
                ((short3*)dst.ptr(y))[x].z += static_cast<short>(v.z * w);
                dst_weight.ptr(y)[x] += w;
            }
        }

        void addSrcWeightGpu32F(const PtrStep<short> src, const PtrStepf src_weight,
            PtrStep<short> dst, PtrStepf dst_weight, cv::Rect &rc)
        {
            dim3 threads(16, 16);
            dim3 grid(divUp(rc.width, threads.x), divUp(rc.height, threads.y));
            addSrcWeightKernel32F<<<grid, threads>>>(src, src_weight, dst, dst_weight, rc.height, rc.width);
            cudaSafeCall(hipGetLastError());
        }

        __global__ void normalizeUsingWeightKernel16S(const PtrStep<short> weight, PtrStep<short> src,
            const int width, const int height)
        {
            int x = (blockIdx.x * blockDim.x) + threadIdx.x;
            int y = (blockIdx.y * blockDim.y) + threadIdx.y;

            if (x < width && y < height)
            {
                const short3 v = ((short3*)src.ptr(y))[x];
                short w = weight.ptr(y)[x];
                ((short3*)src.ptr(y))[x] = make_short3(short((v.x << 8) / w),
                    short((v.y << 8) / w), short((v.z << 8) / w));
            }
        }

        void normalizeUsingWeightMapGpu16S(const PtrStep<short> weight, PtrStep<short> src,
                                           const int width, const int height)
        {
            dim3 threads(16, 16);
            dim3 grid(divUp(width, threads.x), divUp(height, threads.y));
            normalizeUsingWeightKernel16S<<<grid, threads>>> (weight, src, width, height);
        }

        __global__ void normalizeUsingWeightKernel32F(const PtrStepf weight, PtrStep<short> src,
            const int width, const int height)
        {
            int x = (blockIdx.x * blockDim.x) + threadIdx.x;
            int y = (blockIdx.y * blockDim.y) + threadIdx.y;

            if (x < width && y < height)
            {
                const float WEIGHT_EPS = 1e-5f;
                const short3 v = ((short3*)src.ptr(y))[x];
                float w = weight.ptr(y)[x];
                ((short3*)src.ptr(y))[x] = make_short3(static_cast<short>(v.x / (w + WEIGHT_EPS)),
                    static_cast<short>(v.y / (w + WEIGHT_EPS)),
                    static_cast<short>(v.z / (w + WEIGHT_EPS)));
            }
        }

        void normalizeUsingWeightMapGpu32F(const PtrStepf weight, PtrStep<short> src,
                                           const int width, const int height)
        {
            dim3 threads(16, 16);
            dim3 grid(divUp(width, threads.x), divUp(height, threads.y));
            normalizeUsingWeightKernel32F<<<grid, threads>>> (weight, src, width, height);
        }
    }
}}}

#endif
