#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "vector_traits.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <std::size_t N>
        __global__ void fp32_to_fp16(Span<__half> output, View<float> input) {
            using output_vector_type = get_vector_type_t<__half, N>;
            using input_vector_type = get_vector_type_t<float, N>;

            auto output_vPtr = output_vector_type::get_pointer(output.data());
            auto input_vPtr = input_vector_type::get_pointer(input.data());

            for (auto i : grid_stride_range(output.size() / output_vector_type::size())) {
                input_vector_type in_vec;
                v_load(in_vec, input_vPtr[i]);

                output_vector_type out_vec;
                for (int j = 0; j < output_vector_type::size(); j++)
                    out_vec.data[j] = __float2half(in_vec.data[j]);

                v_store(output_vPtr[i], out_vec);
            }
        }

        template <std::size_t N>
        __global__ void fp16_to_fp32(Span<float> output, View<__half> input) {
            using output_vector_type = get_vector_type_t<float, N>;
            using input_vector_type = get_vector_type_t<__half, N>;

            auto output_vPtr = output_vector_type::get_pointer(output.data());
            auto input_vPtr = input_vector_type::get_pointer(input.data());

            for (auto i : grid_stride_range(output.size() / output_vector_type::size())) {
                input_vector_type in_vec;
                v_load(in_vec, input_vPtr[i]);

                output_vector_type out_vec;
                for (int j = 0; j < output_vector_type::size(); j++)
                    out_vec.data[j] = __half2float(in_vec.data[j]);

                v_store(output_vPtr[i], out_vec);
            }
        }
    }

    template <std::size_t N> static
    void launch_vectorized_fp32_to_fp16(const Stream& stream, Span<__half> output, View<float> input) {
        CV_Assert(is_fully_aligned<__half>(output, N));
        CV_Assert(is_fully_aligned<float>(input, N));

        auto kernel = raw::fp32_to_fp16<N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, input);
    }

    void fp32_to_fp16(const Stream& stream, Span<__half> output, View<float> input) {
        if (is_fully_aligned<__half>(output, 4) && is_fully_aligned<float>(input, 4)) {
            launch_vectorized_fp32_to_fp16<4>(stream, output, input);
        } else if (is_fully_aligned<__half>(output, 2) && is_fully_aligned<float>(input, 2)) {
            launch_vectorized_fp32_to_fp16<2>(stream, output, input);
        } else {
            launch_vectorized_fp32_to_fp16<1>(stream, output, input);
        }
    }

    template <std::size_t N> static
    void launch_vectorized_fp16_to_fp32(const Stream& stream, Span<float> output, View<__half> input) {
        CV_Assert(is_fully_aligned<float>(output, N));
        CV_Assert(is_fully_aligned<__half>(input, N));

        auto kernel = raw::fp16_to_fp32<N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, input);
    }

    void fp16_to_fp32(const Stream& stream, Span<float> output, View<__half> input) {
        if (is_fully_aligned<float>(output, 4) && is_fully_aligned<__half>(input, 4)) {
            launch_vectorized_fp16_to_fp32<4>(stream, output, input);
        } else if (is_fully_aligned<float>(output, 2) && is_fully_aligned<__half>(input, 2)) {
            launch_vectorized_fp16_to_fp32<2>(stream, output, input);
        } else {
            launch_vectorized_fp16_to_fp32<1>(stream, output, input);
        }
    }

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
