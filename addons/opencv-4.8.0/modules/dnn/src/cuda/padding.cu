#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "array.hpp"
#include "math.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "kernel_dispatcher.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

#include <cstddef>
#include <vector>
#include <utility>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, std::size_t Rank>
        __global__ void copy_with_reflection101(
            Span<T> output, array<size_type, Rank> out_strides, array<index_type, Rank> start, array<index_type, Rank> end,
            View<T> input, array<size_type, Rank> in_strides)
        {
            for (auto i : grid_stride_range(output.size())) {
                /* compute output axis indices corresponding to element 'i' */
                array<index_type, Rank> out_index;
                out_index[0] = i / out_strides[0];
                for (int j = 1; j < Rank; j++)
                    out_index[j] = (i % out_strides[j - 1]) / out_strides[j];

                /* compute input axis indices corresponding to output axis indices */
                array<index_type, Rank> in_index;
                for (int j = 0; j < Rank; j++) {
                    /* if out_index < start, the point is in the left reflection region
                     * the reflected value's index is the absolute value of the difference
                     *
                     * otherwise, if the value is in the copy region, out_index - start gives the input index
                     */
                    using device::abs;
                    in_index[j] = abs(out_index[j] - start[j]);

                    /* if out_index >= end, it's in the right reflection region */
                    if (out_index[j] >= end[j])
                        in_index[j] = (end[j] - start[j]) - (out_index[j] - end[j]) - 2;
                }

                /* compute input element number from input axis indices */
                index_type iidx = 0;
                for (int j = 0; j < Rank; j++)
                    iidx += in_index[j] * in_strides[j];

                output[i] = input[iidx];
            }
        }
    }

    template <class T, std::size_t Rank> static
    void launch_copy_with_reflection101(
        const Stream& stream,
        Span<T> output, const std::vector<std::size_t>& outStride,
        View<T> input, const std::vector<std::size_t>& inStride,
        const std::vector<std::pair<std::size_t, std::size_t>>& ranges)
    {
        CV_Assert(outStride.size() == Rank);
        CV_Assert(inStride.size() == Rank);
        CV_Assert(ranges.size() == Rank);

        array<size_type, Rank> outStride_k, inStride_k;
        outStride_k.assign(std::begin(outStride), std::end(outStride));
        inStride_k.assign(std::begin(inStride), std::end(inStride));

        array<index_type, Rank> start_k, end_k;
        for (int i = 0; i < Rank; i++) {
            start_k[i] = ranges[i].first;
            end_k[i] = ranges[i].second;
        }

        auto kernel = raw::copy_with_reflection101<T, Rank>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, outStride_k, start_k, end_k, input, inStride_k);
    }

    GENERATE_KERNEL_DISPATCHER(copy_with_reflection101_dispatcher, launch_copy_with_reflection101);

    template <class T>
    void copy_with_reflection101(
        const Stream& stream,
        TensorSpan<T> output, TensorView<T> input,
        std::vector<std::pair<std::size_t, std::size_t>> ranges)
    {
        CV_Assert(output.rank() == input.rank());
        CV_Assert(output.rank() == ranges.size());

        /* squeezable axes at the beginning of both tensors can be eliminated
         *
         * Reasoning:
         * ----------
         * Suppose an item's indices in the input tensor is [i1, i2, ...]. The indices in the
         * output tensor will be [i1 + off1, i2 + off2, ...]. The rest of the elements in the output are padding.
         * The padding operation essentially copies items from the input tensor to new locations in the output tensor
         * and pads the remaining.
         *
         * If the size of the first axis of the input and output tensor is unity, the input and output indices
         * for all the elements will be of the form be [0, i2, ...] and [0, i2 + off2, ...] respectively. Note that
         * there cannot be extra padding since the axes have unit size. The first index does not contribute to the
         * element's address calculation and hence does nothing apart from eating up few cycles.
         */
        while (input.get_axis_size(0) == 1 && output.get_axis_size(0) == 1) {
            CV_Assert(ranges[0].first == 0 && ranges[0].second == 1);

            input.squeeze(0);
            output.squeeze(0);
            ranges.erase(std::begin(ranges));

            CV_Assert(output.rank() == input.rank());
            CV_Assert(output.rank() == ranges.size());
        }

        auto inShape = input.shape_as_vector();
        auto outShape = output.shape_as_vector();

        /* contiguous axes which do not have any padding can be combined into one axis
         *
         * Reasoning:
         * ----------
         * Suppose an item's indices in the input tensor is [i1, i2, i3, ...]. Let the first two axes not have any
         * padding. The indices in the output tensor will be [i1, i2, i3 + off3, ...].
         *
         * Each axis in the contiguous unpadded axes sequence will add an offset of iN * strideN. In the above example,
         * the two axes add a total offset of `i1 * stride1 + i2 * stride2`. We can merge the two axes into one axis with
         * a size of `size1 * size2`. The new offset added will be `i12 * stride2` as the kernel iterates through `i12`.
         * Note that `i12` is actually `(i1 * size2 + i2)` in the original tensor.
         */
        for (int i = 0; i < inShape.size(); i++) {
            /* check if axis `i` requires any padding */
            if (ranges[i].first == 0 && ranges[i].second == inShape[i]) {
                /* loop invariant: `i` is the first axis in the contiguous unpadded axis sequence */
                CV_Assert(inShape[i] == outShape[i]);

                /* we now iterate through the axes which follow and try to merge */
                int j = i + 1; /* `j` is the axis which we will attempt to merge */
                while (j < inShape.size() && ranges[j].first == 0 && ranges[j].second == inShape[j]) {
                    CV_Assert(inShape[j] == outShape[j]);

                    /* `j` is also unpadded; merge `i` and `j` */
                    auto new_size = inShape[i] * inShape[j];
                    inShape[i] = new_size;
                    outShape[i] = new_size;
                    ranges[i].second = new_size;

                    /* delete axis `j` */
                    inShape.erase(std::begin(inShape) + j);
                    outShape.erase(std::begin(outShape) + j);
                    ranges.erase(std::begin(ranges) + j);

                    /* optimizations should not break the invariants */
                    CV_Assert(inShape.size() == outShape.size());
                    CV_Assert(inShape.size() == ranges.size());
                    CV_Assert(inShape[i] == outShape[i]);
                    CV_Assert(ranges[i].first == 0 && ranges[i].second == inShape[i]);
                }
            }
        }

        auto rank = inShape.size();

        std::vector<std::size_t> inStride(rank), outStride(rank);
        inStride.back() = 1;
        outStride.back() = 1;
        /* garbage, ..., garbage, 1 */

        std::copy(std::begin(inShape) + 1, std::end(inShape), std::begin(inStride));
        std::copy(std::begin(outShape) + 1, std::end(outShape), std::begin(outStride));
        /* dim[0], dim[1], ..., dim[-1], 1 */

        std::partial_sum(inStride.rbegin(), inStride.rend(), inStride.rbegin(), std::multiplies<int>());
        std::partial_sum(outStride.rbegin(), outStride.rend(), outStride.rbegin(), std::multiplies<int>());
        /* stride[0], stride[1], ..., stride[-2], 1 */

        CV_Assert(1 <= rank && rank <= CSL_MAX_TENSOR_RANK);
        copy_with_reflection101_dispatcher<T, 1, CSL_MAX_TENSOR_RANK>(rank, stream, output, outStride, input, inStride, ranges);
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void copy_with_reflection101(const Stream&, TensorSpan<__half>, TensorView<__half>, std::vector<std::pair<std::size_t, std::size_t>> ranges);
#endif
    template void copy_with_reflection101(const Stream&, TensorSpan<float>, TensorView<float>, std::vector<std::pair<std::size_t, std::size_t>> ranges);

}}}} /* namespace namespace cv::dnn::cuda4dnn::kernels */
