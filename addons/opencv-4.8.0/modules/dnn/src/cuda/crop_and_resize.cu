#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "memory.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

#include <hip/hip_runtime.h>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {

        template <class T, std::size_t CHANNELS_PER_ITER>
        __global__ void crop_and_resize(
            Span<T> output, size_type out_height, size_type out_width,
            View<T> input, size_type in_height, size_type in_width,
            View<T> boxes,
            size_type num_channels)
        {
            // input [1, num_channels, in_height, in_width]
            // output [boxes, num_channels, out_height, out_width]

            const auto in_image_size = in_height * in_width;
            const auto out_image_size = out_height * out_width;
            const auto out_box_size = num_channels * out_image_size;

            /* we have to compute the output value for every combination of (box, c, y, x) in the output
             *
             * the computation involving (y, x) are identical for all non-spatial dimensions
             * the computation and memory requests involving the box are identical for remaining three axes
             *
             * we process multiple channels every iteration to reuse the identical computation
             * and memory requests involved with the box and spatial dimensions
             */

            /*
             * if we are processing `CHANNELS_PER_ITER` channels per iteration, we will need
             * (num_channels / CHANNELS_PER_ITER) iterations per (box, x, y)
             */
            auto num_channel_iters_per_box_xy = num_channels / CHANNELS_PER_ITER;

            /* we need `num_channel_iters_per_box_xy` iterations per (box, x, y) and there are
             * `num_boxes` boxes and `out_image_size` combinations of (x, y)
             */
            auto num_boxes = boxes.size() / 7; /* 7 values per box */
            auto iters_per_box = num_channel_iters_per_box_xy * out_image_size;
            auto iters_required = num_boxes * iters_per_box;

            for (auto iter : grid_stride_range(iters_required)) {
                const index_type box_no = iter / iters_per_box;
                const index_type c_start = ((iter % iters_per_box) / out_image_size) * CHANNELS_PER_ITER;

                /* note here that consecutive `iter` values will often have consecutive `x` values
                 * => stores into output will be coalesced across threads
                 */
                const index_type y = (iter % out_image_size) / out_width;
                const index_type x = iter % out_width;

                const index_type box_offset = box_no * 7;
                const auto left = boxes[box_offset + 3],
                           top = boxes[box_offset + 4],
                           right = boxes[box_offset + 5],
                           bottom = boxes[box_offset + 6];

                const auto box_width = right - left;
                const auto box_height = bottom - top;

                const auto o2i_fy = static_cast<T>(in_height - 1) / static_cast<T>(out_height - 1);
                const auto o2i_fx = static_cast<T>(in_width - 1) / static_cast<T>(out_width - 1);

                const auto height_scale = box_height * o2i_fy;
                const auto width_scale = box_width * o2i_fx;

                const auto in_y = top * static_cast<T>(in_height - 1) + static_cast<T>(y) * height_scale;
                const auto in_x = left * static_cast<T>(in_width - 1) + static_cast<T>(x) * width_scale;

                const auto in_y0 = static_cast<index_type>(in_y);
                const auto in_x0 = static_cast<index_type>(in_x);

                using device::min;
                const auto in_x1 = min<index_type>(in_x0 + 1, in_width - 1);
                const auto in_y1 = min<index_type>(in_y0 + 1, in_height - 1);

                index_type in_offset_r0 = c_start * in_image_size + in_y0 * in_width;
                index_type in_offset_r1 = c_start * in_image_size + in_y1 * in_width;
                index_type out_idx = box_no * out_box_size + c_start * out_image_size + y * out_width + x;

                #pragma unroll 1 /* disable unrolling */
                for (int i = 0; i < CHANNELS_PER_ITER; i++) {
                    auto v_00 = load_ldg(input[in_offset_r0 + in_x0]),
                         v_01 = load_ldg(input[in_offset_r0 + in_x1]),
                         v_10 = load_ldg(input[in_offset_r1 + in_x0]),
                         v_11 = load_ldg(input[in_offset_r1 + in_x1]);

                    output[out_idx] =
                        v_00 +
                        T(in_y - T(in_y0)) * T(v_10 - v_00) +
                        T(in_x - T(in_x0)) * T(v_01 - v_00) +
                        T(in_y - T(in_y0)) * T(in_x - T(in_x0)) * T(v_11 - v_01 - v_10 + v_00);

                    in_offset_r0 += in_image_size;
                    in_offset_r1 += in_image_size;
                    out_idx += out_image_size;
                }
            }
        }
    }

    template <class T, std::size_t CHANNELS_PER_ITER> static
    void launch_multichannel_crop_and_resize(const Stream& stream,
            Span<T> output, size_type out_height, size_type out_width,
            View<T> input, size_type in_height, size_type in_width,
            View<T> boxes, size_type num_channels)
    {
        auto kernel = raw::crop_and_resize<T, CHANNELS_PER_ITER>;
        auto policy = make_policy(kernel, output.size() / CHANNELS_PER_ITER, 0, stream);
        launch_kernel(kernel, policy, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
    }

    template <class T>
    void crop_and_resize(const Stream& stream, TensorSpan<T> output, TensorView<T> input, View<T> boxes) {
        CV_Assert(input.get_axis_size(0) == 1); /* batch not supported */
        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        auto out_height = output.get_axis_size(-2);
        auto out_width = output.get_axis_size(-1);

        auto in_height = input.get_axis_size(-2);
        auto in_width = input.get_axis_size(-1);

        auto num_channels = input.get_axis_size(1);

        if (num_channels % 64 == 0) {
            launch_multichannel_crop_and_resize<T, 64>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        } else if (num_channels % 32 == 0) {
            launch_multichannel_crop_and_resize<T, 32>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        } else if (num_channels % 16 == 0) {
            launch_multichannel_crop_and_resize<T, 16>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        } else if (num_channels % 8 == 0) {
            launch_multichannel_crop_and_resize<T, 8>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        } else if (num_channels % 4 == 0) {
            launch_multichannel_crop_and_resize<T, 4>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        } else if (num_channels % 2 == 0) {
            launch_multichannel_crop_and_resize<T, 2>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        } else {
            launch_multichannel_crop_and_resize<T, 1>(stream, output, out_height, out_width, input, in_height, in_width, boxes, num_channels);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void crop_and_resize<__half>(const Stream&, TensorSpan<__half>, TensorView<__half>, View<__half> boxes);
#endif
    template void crop_and_resize<float>(const Stream&, TensorSpan<float>, TensorView<float>, View<float> boxes);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
