#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "functors.hpp"
#include "types.hpp"
#include "vector_traits.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

namespace raw {

    template <class T, class ActivationOp, class EltwiseOp, std::size_t N>
    __global__ void biasN_generic_op_eltwise_op_inplace_vec(Span<T> inplace_output, size_type inner_size, View<T> bias, View<T> eltwise, const typename ActivationOp::Params act_params, const typename EltwiseOp::Params eltwise_params) {
        using vector_type = get_vector_type_t<T, N>;

        auto inplace_output_vPtr = vector_type::get_pointer(inplace_output.data());
        auto eltwise_vPtr = vector_type::get_pointer(eltwise.data());

        ActivationOp activation_op(act_params);
        EltwiseOp eltwise_op(eltwise_params);

        for (auto i : grid_stride_range(inplace_output.size() / vector_type::size())) {
            const index_type bias_idx = (i / inner_size) % bias.size();

            vector_type output_vec, eltwise_vec;
            v_load(output_vec, inplace_output_vPtr[i]);
            v_load(eltwise_vec, eltwise_vPtr[i]);
            for(int j = 0; j < output_vec.size(); j++)
                output_vec.data[j] = eltwise_op(activation_op(output_vec.data[j] + bias[bias_idx]), eltwise_vec.data[j]);
            v_store(inplace_output_vPtr[i], output_vec);
        }
    }
}

template <class T, class ActivationOp, class EltwiseOp, std::size_t N> static
void launch_vectorized_biasN_generic_op_eltwise_op_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, const typename ActivationOp::Params& act_params, const typename EltwiseOp::Params& eltwise_params) {
    CV_Assert(is_fully_aligned<T>(inplace_output, N));
    CV_Assert(is_fully_aligned<T>(eltwise, N));
    CV_Assert(inner_size % N == 0);

    auto kernel = raw::biasN_generic_op_eltwise_op_inplace_vec<T, ActivationOp, EltwiseOp, N>;
    auto policy = make_policy(kernel, inplace_output.size() / N, 0, stream);
    launch_kernel(kernel, policy, inplace_output, inner_size / N, bias, eltwise, act_params, eltwise_params);
}

template <class T, class ActivationOp, class EltwiseOp> static
void biasN_generic_op_eltwise_op_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, const typename ActivationOp::Params& act_params = {}, const typename EltwiseOp::Params& eltwise_params = {}) {
    CV_Assert(inplace_output.size() == eltwise.size());

    if (is_fully_aligned<T>(inplace_output, 4) && is_fully_aligned<T>(eltwise, 4) && inner_size % 4 == 0) {
        launch_vectorized_biasN_generic_op_eltwise_op_inplace<T, ActivationOp, EltwiseOp, 4>(stream, inplace_output, inner_size, bias, eltwise, act_params, eltwise_params);
    } else if (is_fully_aligned<T>(inplace_output, 2) && is_fully_aligned<T>(eltwise, 2) && inner_size % 2 == 0) {
        launch_vectorized_biasN_generic_op_eltwise_op_inplace<T, ActivationOp, EltwiseOp, 2>(stream, inplace_output, inner_size, bias, eltwise, act_params, eltwise_params);
    } else {
        launch_vectorized_biasN_generic_op_eltwise_op_inplace<T, ActivationOp, EltwiseOp, 1>(stream, inplace_output, inner_size, bias, eltwise, act_params, eltwise_params);
    }
}

template <class T>
void biasN_relu_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, T slope) {
    biasN_generic_op_eltwise_op_inplace<T, ReLUFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise, {slope});
}

template <class T>
void biasN_clipped_relu_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, T floor, T ceiling) {
    CV_Assert(static_cast<double>(floor) <= static_cast<double>(ceiling));
    biasN_generic_op_eltwise_op_inplace<T, ClippedReLUFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise, {floor, ceiling});
}

template <class T>
void biasN_tanh_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_generic_op_eltwise_op_inplace<T, TanHFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_swish_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_generic_op_eltwise_op_inplace<T, SwishFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_mish_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_generic_op_eltwise_op_inplace<T, MishFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_sigmoid_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_generic_op_eltwise_op_inplace<T, SigmoidFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_power_eltwise_sum_2_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, T exp, T scale, T shift) {
    biasN_generic_op_eltwise_op_inplace<T, PowerFunctor<T>, SumFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise, {exp, scale, shift});
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void biasN_relu_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>, __half);
template void biasN_clipped_relu_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>, __half, __half);
template void biasN_tanh_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_swish_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_mish_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_sigmoid_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_power_eltwise_sum_2_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>, __half, __half, __half);
#endif

template void biasN_relu_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>, float);
template void biasN_clipped_relu_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>, float, float);
template void biasN_tanh_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_swish_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_mish_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_sigmoid_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_power_eltwise_sum_2_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>, float, float, float);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
