#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "functors.hpp"
#include "types.hpp"
#include "vector_traits.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

namespace raw {
    template <class T, class EltwiseOp, class ActivationOp, std::size_t N>
    __global__ void biasN_eltwise_op_generic_op_inplace_vec(Span<T> inplace_output, size_type inner_size, View<T> bias, View<T> eltwise, const typename EltwiseOp::Params eltwise_params, const typename ActivationOp::Params act_params) {
        using vector_type = get_vector_type_t<T, N>;

        auto inplace_output_vPtr = vector_type::get_pointer(inplace_output.data());
        auto eltwise_vPtr = vector_type::get_pointer(eltwise.data());

        EltwiseOp eltwise_op(eltwise_params);
        ActivationOp activation_op(act_params);

        for (auto i : grid_stride_range(inplace_output.size() / vector_type::size())) {
            const index_type bias_idx = (i / inner_size) % bias.size();

            vector_type output_vec, eltwise_vec;
            v_load(output_vec, inplace_output_vPtr[i]);
            v_load(eltwise_vec, eltwise_vPtr[i]);
            for(int j = 0; j < output_vec.size(); j++)
                output_vec.data[j] = activation_op(eltwise_op(output_vec.data[j] + bias[bias_idx], eltwise_vec.data[j]));
            v_store(inplace_output_vPtr[i], output_vec);
        }
    }
}

template <class T, class EltwiseOp, class ActivationOp, std::size_t N> static
void launch_vectorized_biasN_eltwise_op_generic_op_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, const typename EltwiseOp::Params& eltwise_params, const typename ActivationOp::Params& act_params) {
    CV_Assert(is_fully_aligned<T>(inplace_output, N));
    CV_Assert(inplace_output.size() % bias.size() == 0);
    CV_Assert(is_fully_aligned<T>(eltwise, N));
    CV_Assert(inner_size % N == 0);

    auto kernel = raw::biasN_eltwise_op_generic_op_inplace_vec<T, EltwiseOp, ActivationOp, N>;
    auto policy = make_policy(kernel, inplace_output.size() / N, 0, stream);
    launch_kernel(kernel, policy, inplace_output, inner_size / N, bias, eltwise, eltwise_params, act_params);
}

template <class T, class EltwiseOp, class ActivationOp> static
void biasN_eltwise_op_generic_op_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, const typename EltwiseOp::Params& eltwise_params = {}, const typename ActivationOp::Params& act_params = {}) {
    CV_Assert(inplace_output.size() == eltwise.size());

    if (is_fully_aligned<T>(inplace_output, 4) && is_fully_aligned<T>(eltwise, 4) && inner_size % 4 == 0) {
        launch_vectorized_biasN_eltwise_op_generic_op_inplace<T, EltwiseOp, ActivationOp, 4>(stream, inplace_output, inner_size, bias, eltwise, eltwise_params, act_params);
    } else if (is_fully_aligned<T>(inplace_output, 2) && is_fully_aligned<T>(eltwise, 2) && inner_size % 2 == 0) {
        launch_vectorized_biasN_eltwise_op_generic_op_inplace<T, EltwiseOp, ActivationOp, 2>(stream, inplace_output, inner_size, bias, eltwise, eltwise_params, act_params);
    } else {
        launch_vectorized_biasN_eltwise_op_generic_op_inplace<T, EltwiseOp, ActivationOp, 1>(stream, inplace_output, inner_size, bias, eltwise, eltwise_params, act_params);
    }
}

template <class T>
void biasN_eltwise_sum_2_identity_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, IdentityFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_eltwise_sum_2_relu_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, T slope) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, ReLUFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise, {}, {slope});
}

template <class T>
void biasN_eltwise_sum_2_clipped_relu_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, T floor, T ceiling) {
    CV_Assert(static_cast<double>(floor) <= static_cast<double>(ceiling));
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, ClippedReLUFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise, {}, {floor, ceiling});
}

template <class T>
void biasN_eltwise_sum_2_tanh_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, TanHFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_eltwise_sum_2_swish_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, SwishFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_eltwise_sum_2_mish_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, MishFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_eltwise_sum_2_sigmoid_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, SigmoidFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise);
}

template <class T>
void biasN_eltwise_sum_2_power_inplace(const Stream& stream, Span<T> inplace_output, std::size_t inner_size, View<T> bias, View<T> eltwise, T exp, T scale, T shift) {
    biasN_eltwise_op_generic_op_inplace<T, SumFunctor<T>, PowerFunctor<T>>(stream, inplace_output, inner_size, bias, eltwise, {}, {exp, scale, shift});
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void biasN_eltwise_sum_2_identity_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_eltwise_sum_2_relu_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>, __half);
template void biasN_eltwise_sum_2_clipped_relu_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>, __half, __half);
template void biasN_eltwise_sum_2_tanh_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_eltwise_sum_2_swish_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_eltwise_sum_2_mish_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_eltwise_sum_2_sigmoid_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>);
template void biasN_eltwise_sum_2_power_inplace<__half>(const Stream&, Span<__half>, std::size_t, View<__half>, View<__half>, __half, __half, __half);
#endif

template void biasN_eltwise_sum_2_identity_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_eltwise_sum_2_relu_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>, float);
template void biasN_eltwise_sum_2_clipped_relu_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>, float, float);
template void biasN_eltwise_sum_2_tanh_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_eltwise_sum_2_swish_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_eltwise_sum_2_mish_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_eltwise_sum_2_sigmoid_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>);
template void biasN_eltwise_sum_2_power_inplace<float>(const Stream&, Span<float>, std::size_t, View<float>, View<float>, float, float, float);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
