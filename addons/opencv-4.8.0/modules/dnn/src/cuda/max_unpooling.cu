#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "array.hpp"
#include "limits.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include "../cuda4dnn/kernels/fill_copy.hpp"

#include <opencv2/core.hpp>

#include <cstddef>
#include <vector>
#include <type_traits>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, std::size_t Order,
        typename std::enable_if<Order == 1 || Order == 2 || Order == 3, bool>::type = true> /* Order has been hardcoded; see code */
        __global__ void max_pooling_with_indices(
            Span<T> output, Span<T> indices, View<T> input, size_type channels,
            array<size_type, Order> out_spatial_dims, array<size_type, Order> in_spatial_dims,
            array<size_type, Order> window_size, array<size_type, Order> strides, array<size_type, Order> padding_left)
        {
            /* every element in the output is mapped to a window in the input and each thread processes several windows */
            for (auto idx : grid_stride_range(output.size())) {
                size_type out_spatial_size = 1;
                array<index_type, Order> window_idx;
                for (int i = Order - 1; i >= 0; i--) {
                    window_idx[i] = (idx / out_spatial_size) % out_spatial_dims[i];
                    out_spatial_size *= out_spatial_dims[i];
                }

                const index_type n = idx / (out_spatial_size * channels);
                const index_type c = (idx / out_spatial_size) % channels;

                array<index_type, Order> start;
                for(int i = 0; i < Order; i++)
                    start[i] = window_idx[i] * strides[i] - padding_left[i];

                array<index_type, Order> end;
                for (int i = 0; i < Order; i++) {
                    using device::min;
                    end[i] = min<index_type>(start[i] + window_size[i], in_spatial_dims[i]);
                }

                for (int i = 0; i < Order; i++) {
                    using device::max;
                    start[i] = max(start[i], 0);
                }

                T max_value = numeric_limits<T>::lowest();
                index_type max_idx = -1;

                size_type in_spatial_size = 1;
                for (int i = 0; i < Order; i++)
                    in_spatial_size *= in_spatial_dims[i];

                const auto outer_offset =  (n * channels + c) * in_spatial_size;
                if (Order == 1) {
                    array<index_type, Order> idx;
                    for (idx[0] = start[0]; idx[0] != end[0]; idx[0]++) {
                        index_type offset = 0;
                        index_type stride = 1;
                        for (int i = Order - 1; i >= 0; i--) {
                            offset += stride * idx[i];
                            stride *= in_spatial_dims[i];
                        }

                        if (input[outer_offset + offset] > max_value) {
                            max_idx = offset;
                            max_value = input[outer_offset + offset];
                        }
                    }
                } else if (Order == 2) {
                    array<index_type, Order> idx;
                    for (idx[0] = start[0]; idx[0] != end[0]; idx[0]++) {
                        for (idx[1] = start[1]; idx[1] != end[1]; idx[1]++) {
                            index_type offset = 0;
                            index_type stride = 1;
                            for (int i = Order - 1; i >= 0; i--) {
                                offset += stride * idx[i];
                                stride *= in_spatial_dims[i];
                            }

                            if (input[outer_offset + offset] > max_value) {
                                max_idx = offset;
                                max_value = input[outer_offset + offset];
                            }
                        }
                    }
                } else if(Order == 3) {
                    array<index_type, Order> idx;
                    for (idx[0] = start[0]; idx[0] != end[0]; idx[0]++) {
                        for (idx[1] = start[1]; idx[1] != end[1]; idx[1]++) {
                            for (idx[2] = start[2]; idx[2] != end[2]; idx[2]++) {
                                index_type offset = 0;
                                index_type stride = 1;
                                for (int i = Order - 1; i >= 0; i--) {
                                    offset += stride * idx[i];
                                    stride *= in_spatial_dims[i];
                                }

                                if (input[outer_offset + offset] > max_value) {
                                    max_idx = offset;
                                    max_value = input[outer_offset + offset];
                                }
                            }
                        }
                    }
                }

                output[idx] = max_value;
                indices[idx] = max_idx;
            }
        }

        template <class T, std::size_t Order>
        __global__ void max_unpooling(
            Span<T> output, View<T> input, View<T> indices, size_type channels,
            array<size_type, Order> out_spatial_dims, array<size_type, Order> in_spatial_dims,
            array<size_type, Order> window_size, array<size_type, Order> strides, array<size_type, Order> padding_left)
        {
            /* the output has already been zero filled */
            /* Every input value represents a window in the output. The max unpooling operation
             * copies the input value to exactly one location in the output window which is given
             * by the indices tensor.
             */
            for (auto idx : grid_stride_range(input.size())) {
                size_type in_spatial_size = 1;
                array<index_type, Order> window_idx;
                for (int i = Order - 1; i >= 0; i--) {
                    window_idx[i] = (idx / in_spatial_size) % in_spatial_dims[i];
                    in_spatial_size *= in_spatial_dims[i];
                }

                const index_type n = idx / (in_spatial_size * channels);
                const index_type c = (idx / in_spatial_size) % channels;

                array<index_type, Order> start;
                for (int i = 0; i < Order; i++) {
                    using device::min;
                    using device::max;
                    start[i] = max(0, min(window_idx[i] * strides[i] - padding_left[i], out_spatial_dims[i] - 1));
                }

                size_type out_spatial_size = 1;
                for (int i = 0; i < Order; i++)
                    out_spatial_size *= out_spatial_dims[i];

                index_type outer_offset = (n * channels + c) * out_spatial_size;
                output[outer_offset + static_cast<index_type>(indices[idx])] = input[idx];
            }
        }
    }

    template <class T, std::size_t Order> static
    void launch_max_pooling_kernel(
        const Stream& stream,
        Span<T> output, Span<T> indices, View<T> input, std::size_t channels,
        const std::vector<std::size_t>& out_spatial_dims, const std::vector<std::size_t>& in_spatial_dims,
        const std::vector<std::size_t>& window_size,
        const std::vector<std::size_t>& strides, const std::vector<std::size_t>& padding_left)
    {
        CV_Assert(indices.size() == output.size());
        CV_Assert(out_spatial_dims.size() == Order);
        CV_Assert(in_spatial_dims.size() == Order);
        CV_Assert(window_size.size() == Order);
        CV_Assert(strides.size() == Order);
        CV_Assert(padding_left.size() == Order);

        array<size_type, Order> out_spatial_dims_k, in_spatial_dims_k;
        out_spatial_dims_k.assign(std::begin(out_spatial_dims), std::end(out_spatial_dims));
        in_spatial_dims_k.assign(std::begin(in_spatial_dims), std::end(in_spatial_dims));

        array<size_type, Order> window_size_k, strides_k, padding_left_k;
        window_size_k.assign(std::begin(window_size), std::end(window_size));
        strides_k.assign(std::begin(strides), std::end(strides));
        padding_left_k.assign(std::begin(padding_left), std::end(padding_left));

        auto kernel = raw::max_pooling_with_indices<T, Order>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, indices, input, channels,
            out_spatial_dims_k, in_spatial_dims_k, window_size_k, strides_k, padding_left_k);
    }

    template <class T>
    void max_pooling_with_indices(
        const Stream& stream,
        TensorSpan<T> output, TensorSpan<T> indices, TensorView<T> input,
        const std::vector<std::size_t>& window_size, const std::vector<std::size_t>& strides,
        const std::vector<std::size_t>& padding_left)
    {
        CV_Assert(is_shape_same(output, indices));
        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        auto order = window_size.size();
        CV_Assert(strides.size() == order);
        CV_Assert(padding_left.size() == order);
        CV_Assert(output.rank() == order + 2);
        CV_Assert(input.rank() == order + 2);

        std::vector<std::size_t> out_spatial_dims(order), in_spatial_dims(order);
        for (int i = 0; i < order; i++) {
            in_spatial_dims[i] = input.get_axis_size(2 + i);
            out_spatial_dims[i] = output.get_axis_size(2 + i);
        }

        CV_Assert(1 <= order && order <= 3);
        std::size_t channels = input.get_axis_size(1);
        if (order == 3) {
            launch_max_pooling_kernel<T, 3>(stream, output, indices, input, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        } else if (order == 2) {
            launch_max_pooling_kernel<T, 2>(stream, output, indices, input, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        } else if (order == 1) {
            launch_max_pooling_kernel<T, 1>(stream, output, indices, input, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void max_pooling_with_indices(const Stream&,
        TensorSpan<__half>, TensorSpan<__half>, TensorView<__half>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);
#endif

    template void max_pooling_with_indices(const Stream&,
        TensorSpan<float>, TensorSpan<float>, TensorView<float>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

    template <class T, std::size_t Order> static
    void launch_max_unpooling_kernel(
        const Stream& stream,
        Span<T> output, View<T> input, View<T> indices, std::size_t channels,
        const std::vector<std::size_t>& out_spatial_dims, const std::vector<std::size_t>& in_spatial_dims,
        const std::vector<std::size_t>& window_size,
        const std::vector<std::size_t>& strides, const std::vector<std::size_t>& padding_left)
    {
        CV_Assert(out_spatial_dims.size() == Order);
        CV_Assert(in_spatial_dims.size() == Order);
        CV_Assert(window_size.size() == Order);
        CV_Assert(strides.size() == Order);
        CV_Assert(padding_left.size() == Order);
        CV_Assert(indices.size() == input.size());

        array<size_type, Order> out_spatial_dims_k, in_spatial_dims_k;
        out_spatial_dims_k.assign(std::begin(out_spatial_dims), std::end(out_spatial_dims));
        in_spatial_dims_k.assign(std::begin(in_spatial_dims), std::end(in_spatial_dims));

        array<size_type, Order> window_size_k, strides_k, padding_left_k;
        window_size_k.assign(std::begin(window_size), std::end(window_size));
        strides_k.assign(std::begin(strides), std::end(strides));
        padding_left_k.assign(std::begin(padding_left), std::end(padding_left));

        auto kernel = raw::max_unpooling<T, Order>;
        auto policy = make_policy(kernel, input.size(), 0, stream);
        launch_kernel(kernel, policy, output, input, indices, channels,
            out_spatial_dims_k, in_spatial_dims_k, window_size_k, strides_k, padding_left_k);
    }

    template <class T>
    void max_unpooling(
        const Stream& stream,
        TensorSpan<T> output, TensorView<T> input, TensorView<T> indices,
        const std::vector<std::size_t>& window_size, const std::vector<std::size_t>& strides,
        const std::vector<std::size_t>& padding_left)
    {
        CV_Assert(is_shape_same(input, indices));
        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        auto order = window_size.size();
        CV_Assert(strides.size() == order);
        CV_Assert(padding_left.size() == order);
        CV_Assert(output.rank() == order + 2);
        CV_Assert(input.rank() == order + 2);

        std::vector<std::size_t> out_spatial_dims(order), in_spatial_dims(order);
        for (int i = 0; i < order; i++) {
            in_spatial_dims[i] = input.get_axis_size(2 + i);
            out_spatial_dims[i] = output.get_axis_size(2 + i);
        }

        kernels::fill<T>(stream, output, 0.0);

        /* only max_unpooling2d and max_unpooling3d are supported */
        CV_Assert(2 <= order && order <= 3);
        std::size_t channels = input.get_axis_size(1);
        if (order == 3) {
            launch_max_unpooling_kernel<T, 3>(stream, output, input, indices, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        } else if (order == 2) {
            launch_max_unpooling_kernel<T, 2>(stream, output, input, indices, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void max_unpooling(const Stream&,
        TensorSpan<__half>, TensorView<__half>, TensorView<__half>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);
#endif

    template void max_unpooling(const Stream&,
        TensorSpan<float>, TensorView<float>, TensorView<float>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
