#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "memory.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <hip/hip_runtime.h>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, std::size_t CHANNELS_PER_ITER>
        __global__ void resize_nn(
            Span<T> output, size_type out_height, size_type out_width,
            View<T> input, size_type in_height, size_type in_width,
            float o2i_fy, float o2i_fx, bool round, bool half_pixel_centers)
        {
            auto in_image_size = in_height * in_width;
            auto out_image_size = out_height * out_width;

            /* think of the output and input as a collection of 2d images with the last axis
             * representing the width and the last but one axis representing the height
             *
             * the remaining axis together form a collection of these images/channels
             */
            auto num_effective_channels = output.size() / out_image_size;

            /* we process multiple channels every iteration to reuse the identical computation
             * involved with the spatial dimensions
             *
             * if we are processing `CHANNELS_PER_ITER` channels per iteration, we will need
             * (num_effective_channels / CHANNELS_PER_ITER) iterations per (x, y) location
             */
            auto num_channel_iters_per_xy = (num_effective_channels / CHANNELS_PER_ITER);

            /* we need `num_channel_iters_per_xy` iterations per (x, y) and there are `out_image_size`
             * combinations of (x, y); hence, we'll need `num_channel_iters_per_xy * out_image_size`
             * iterations in total to finish the resize operation
             */
            auto iters_required = num_channel_iters_per_xy * out_image_size;
            for (auto iter : grid_stride_range(iters_required)) {
                const index_type c_start = (iter / out_image_size) * CHANNELS_PER_ITER;

                /* note here that consecutive `iter` values will often have consecutive `x` values
                 * => stores into output will be coalesced across threads
                 */
                const index_type y = (iter % out_image_size) / out_width;
                const index_type x = iter % out_width;

                auto in_yf = half_pixel_centers ? (y + 0.5f) * o2i_fy : y * o2i_fy;
                auto in_xf = half_pixel_centers ? (x + 0.5f) * o2i_fx : x * o2i_fx;

                using device::lround;
                index_type in_y = round ? lround(in_yf) : static_cast<index_type>(in_yf);
                index_type in_x = round ? lround(in_xf) : static_cast<index_type>(in_xf);

                using device::min;
                in_y = min(in_y, in_height - 1);
                in_x = min(in_x, in_width - 1);

                index_type in_idx = c_start * in_image_size + in_y * in_width + in_x;
                index_type out_idx = c_start * out_image_size + y * out_width + x;

                for (int i = 0; i < CHANNELS_PER_ITER; i++) {
                    output[out_idx] = load_ldg(input[in_idx]);

                    in_idx += in_image_size;
                    out_idx += out_image_size;
                }
            }
        }

        template <class T, std::size_t CHANNELS_PER_ITER>
        __global__ void resize_bilinear(
            Span<T> output, size_type out_height, size_type out_width,
            View<T> input, size_type in_height, size_type in_width,
            float o2i_fy, float o2i_fx, bool half_pixel_centers)
        {
            auto in_image_size = in_height * in_width;
            auto out_image_size = out_height * out_width;

            /* think of the output and input as a collection of 2d images with the last axis
             * representing the width and the last but one axis representing the height
             *
             * the remaining axis together form a collection of these images/channels
             */
            auto num_effective_channels = output.size() / out_image_size;

            /* we process multiple channels every iteration to reuse the identical computation
             * involved with the spatial dimensions
             *
             * if we are processing `CHANNELS_PER_ITER` channels per iteration, we will need
             * (num_effective_channels / CHANNELS_PER_ITER) iterations per (x, y) location
             */
            auto num_channel_iters_per_xy = (num_effective_channels / CHANNELS_PER_ITER);

            /* we need `num_channel_iters_per_xy` iterations per (x, y) and there are `out_image_size`
             * combinations of (x, y); hence, we'll need `num_channel_iters_per_xy * out_image_size`
             * iterations in total to finish the resize operation
             */
            auto iters_required = num_channel_iters_per_xy * out_image_size;

            for (auto iter : grid_stride_range(iters_required)) {
                const index_type c_start = (iter / out_image_size) * CHANNELS_PER_ITER;
                const index_type c_end = c_start + CHANNELS_PER_ITER;

                /* note here that consecutive `iter` values will often have consecutive `x` values
                 * => stores into output will be coalesced across threads
                 */
                const index_type y = (iter % out_image_size) / out_width;
                const index_type x = iter % out_width;

                using device::max;
                auto in_x = half_pixel_centers ? max<float>((x + 0.5f) * o2i_fx - 0.5f, 0.0f) : x * o2i_fx;
                auto in_y = half_pixel_centers ? max<float>((y + 0.5f) * o2i_fy - 0.5f, 0.0f) : y * o2i_fy;

                auto in_x0 = static_cast<index_type>(in_x);
                auto in_y0 = static_cast<index_type>(in_y);

                using device::min;
                auto in_x1 = min<index_type>(in_x0 + 1, in_width - 1);
                auto in_y1 = min<index_type>(in_y0 + 1, in_height - 1);

                index_type in_offset_r0 = c_start * in_image_size + in_y0 * in_width;
                index_type in_offset_r1 = c_start * in_image_size + in_y1 * in_width;
                index_type out_idx = c_start * out_image_size + y * out_width + x;

                #pragma unroll 1 /* disable unrolling to reduce register pressure; not sure how but it works */
                for (auto c = c_start; c < c_end; c++) {
                    auto v_00 = load_ldg(input[in_offset_r0 + in_x0]),
                         v_01 = load_ldg(input[in_offset_r0 + in_x1]),
                         v_10 = load_ldg(input[in_offset_r1 + in_x0]),
                         v_11 = load_ldg(input[in_offset_r1 + in_x1]);

                    output[out_idx] =
                        v_00 +
                        T(in_y - in_y0) * T(v_10 - v_00) +
                        T(in_x - in_x0) * T(v_01 - v_00) +
                        T(in_y - in_y0) * T(in_x - in_x0) * T(v_11 - v_01 - v_10 + v_00);

                    in_offset_r0 += in_image_size;
                    in_offset_r1 += in_image_size;
                    out_idx += out_image_size;
                }
            }
        }
    }

    template <class T, std::size_t CHANNELS_PER_ITER> static
    void launch_multichannel_resize_nn(const Stream& stream,
        Span<T> output, size_type out_height, size_type out_width,
        View<T> input, size_type in_height, size_type in_width,
        float scale_y, float scale_x, bool round, bool half_pixel_centers)
    {
        auto kernel = raw::resize_nn<T, CHANNELS_PER_ITER>;
        auto policy = make_policy(kernel, output.size() / CHANNELS_PER_ITER, 0, stream);
        launch_kernel(kernel, policy, output, out_height, out_width, input, in_height, in_width,  scale_y, scale_x, round, half_pixel_centers);
    }

    template <class T>
    void resize_nn(const Stream& stream, TensorSpan<T> output, TensorView<T> input, float scale_y, float scale_x, bool round, bool half_pixel_centers) {
        auto out_height = output.get_axis_size(-2);
        auto out_width = output.get_axis_size(-1);

        auto in_height = input.get_axis_size(-2);
        auto in_width = input.get_axis_size(-1);

        auto num_effective_channels = input.size_range(0, 2);
        auto num_iters = num_effective_channels * out_height * out_width;

        if (num_effective_channels % 32 == 0 && num_iters > 655360) {
            launch_multichannel_resize_nn<T, 32>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, round, half_pixel_centers);
        } else if (num_effective_channels % 16 == 0 && num_iters > 327680) {
            launch_multichannel_resize_nn<T, 16>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, round, half_pixel_centers);
        } else if (num_effective_channels % 8 == 0 && num_iters > 163840) {
            launch_multichannel_resize_nn<T, 8>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, round, half_pixel_centers);
        } else if (num_effective_channels % 4 == 0 && num_iters > 81920) {
            launch_multichannel_resize_nn<T, 4>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, round, half_pixel_centers);
        } else if (num_effective_channels % 2 == 0) {
            launch_multichannel_resize_nn<T, 2>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, round, half_pixel_centers);
        } else {
            launch_multichannel_resize_nn<T, 1>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, round, half_pixel_centers);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void resize_nn<__half>(const Stream&, TensorSpan<__half>, TensorView<__half>, float, float, bool, bool);
#endif
    template void resize_nn<float>(const Stream&, TensorSpan<float>, TensorView<float>, float, float, bool,bool);

    template <class T, std::size_t CHANNELS_PER_ITER> static
    void launch_multichannel_resize_bilinear(const Stream& stream,
        Span<T> output, size_type out_height, size_type out_width,
        View<T> input, size_type in_height, size_type in_width,
        float scale_y, float scale_x, bool half_pixel_centers)
    {
        auto kernel = raw::resize_bilinear<T, CHANNELS_PER_ITER>;
        auto policy = make_policy(kernel, output.size() / CHANNELS_PER_ITER, 0, stream);
        launch_kernel(kernel, policy, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, half_pixel_centers);
    }

    template <class T>
    void resize_bilinear(const Stream& stream, TensorSpan<T> output, TensorView<T> input, float scale_y, float scale_x, bool half_pixel_centers) {
        auto out_height = output.get_axis_size(-2);
        auto out_width = output.get_axis_size(-1);

        auto in_height = input.get_axis_size(-2);
        auto in_width = input.get_axis_size(-1);

        auto num_effective_channels = input.size_range(0, 2);
        auto num_iters = num_effective_channels * out_height * out_width;

        if (num_effective_channels % 16 == 0 && num_iters > 163840) {
            launch_multichannel_resize_bilinear<T, 16>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, half_pixel_centers);
        } else if (num_effective_channels % 8 == 0 && num_iters > 81920) {
            launch_multichannel_resize_bilinear<T, 8>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, half_pixel_centers);
        } else if (num_effective_channels % 4 == 0 && num_iters > 40960) {
            launch_multichannel_resize_bilinear<T, 4>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, half_pixel_centers);
        } else if (num_effective_channels % 2 == 0) {
            launch_multichannel_resize_bilinear<T, 2>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, half_pixel_centers);
        } else {
            launch_multichannel_resize_bilinear<T, 1>(stream, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x, half_pixel_centers);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void resize_bilinear<__half>(const Stream&, TensorSpan<__half>, TensorView<__half>, float, float, bool);
#endif
    template void resize_bilinear<float>(const Stream&, TensorSpan<float>, TensorView<float>, float, float, bool);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
