#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "vector_traits.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"
#include "../cuda4dnn/csl/tensor.hpp"

#include <opencv2/core.hpp>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

namespace raw {
    template <class T, std::size_t N>
    __global__ void input_shortcut_vec(
        Span<T> output,
        View<T> input, index_type c_input, /* `c_input` = number of channels in `input` */
        View<T> from, index_type c_from, /* `c_from` = number of channels in `from` */
        size_type channel_stride /* common for both `input` and `from` */)
    {
        using vector_type = get_vector_type_t<T, N>;

        auto output_vPtr = vector_type::get_pointer(output.data());
        auto input_vPtr = vector_type::get_pointer(input.data());
        auto from_vPtr = vector_type::get_pointer(from.data());

        auto batch_stride_input = c_input * channel_stride;
        auto batch_stride_from = c_from * channel_stride;

        for (auto i : grid_stride_range(output.size() / vector_type::size())) {
            const auto actual_idx = i * vector_type::size();
            const auto b = actual_idx / batch_stride_input; /* `input` and `output` have the same shape */
            const auto c = (actual_idx % batch_stride_input) / channel_stride;
            const auto c_offset = actual_idx % channel_stride;

            vector_type vec_input;
            v_load(vec_input, input_vPtr[i]);

            /* We can break down the shortcut operation into two steps:
             * - copy `input` to `output`
             * - add `from` to corresponding channels in `output`
             *
             * In this scheme, only some channels in the `output` differ from `input`. They differ in the channels
             * which have a corresponding channel in `from`.
             */
            if (c < c_from) {
                const auto from_actual_idx = b * batch_stride_from + c * channel_stride + c_offset;
                const auto from_vec_idx = from_actual_idx / vector_type::size();

                vector_type vec_from;
                v_load(vec_from, from_vPtr[from_vec_idx]);
                for (int j = 0; j < vector_type::size(); j++)
                    vec_input.data[j] += vec_from.data[j];
            }

            v_store(output_vPtr[i], vec_input);
        }
    }
}

template <class T, std::size_t N>
void launch_vectorized_input_shortcut(const Stream& stream, Span<T> output, View<T> input, std::size_t c_input, View<T> from, std::size_t c_from, std::size_t channel_stride) {
    CV_Assert(is_fully_aligned<T>(output, N));
    CV_Assert(is_fully_aligned<T>(input, N));
    CV_Assert(is_fully_aligned<T>(from, N));
    CV_Assert(channel_stride % N == 0);

    auto kernel = raw::input_shortcut_vec<T, N>;
    auto policy = make_policy(kernel, output.size() / N, 0, stream);
    launch_kernel(kernel, policy, output, input, c_input, from, c_from, channel_stride);
}

template <class T>
void input_shortcut(const csl::Stream& stream, csl::TensorSpan<T> output, csl::TensorView<T> input, csl::TensorView<T> from) {
    CV_Assert(is_shape_same(output, input));
    CV_Assert(output.rank() == from.rank());
    for (int i = 0; i < output.rank(); i++) {
        if (i != 1) {
            CV_Assert(from.get_axis_size(i) == output.get_axis_size(i));
        }
    }

    auto channel_stride = output.size_range(2, output.rank()); /* same for `output`, `input` and `from` */
    auto c_input = input.get_axis_size(1);
    auto c_from = from.get_axis_size(1);

    if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(input, 4) && is_fully_aligned<T>(from, 4) && channel_stride % 4 == 0) {
        launch_vectorized_input_shortcut<T, 4>(stream, output, input, c_input, from, c_from, channel_stride);
    } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(input, 2) && is_fully_aligned<T>(from, 2) && channel_stride % 2 == 0) {
        launch_vectorized_input_shortcut<T, 2>(stream, output, input, c_input, from, c_from, channel_stride);
    } else {
        launch_vectorized_input_shortcut<T, 1>(stream, output, input, c_input, from, c_from, channel_stride);
    }
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void input_shortcut(const Stream&, TensorSpan<__half>, TensorView<__half>, TensorView<__half>);
#endif
template void input_shortcut(const Stream&, TensorSpan<float>, TensorView<float>, TensorView<float>);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
