#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "limits.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "memory.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {

        template <class T, std::size_t CHANNELS_PER_ITER>
        __global__ void roi_pooling(
            Span<T> output, size_type pooled_height, size_type pooled_width,
            View<T> input, size_type in_height, size_type in_width,
            View<T> rois, size_type num_channels, float spatial_scale)
        {
            // input: [1, num_channels, in_height, in_width]
            const auto in_image_size = in_height * in_width;

            // rois: [num_rois, 5]
            auto num_rois = rois.size() / 5;

            // output: [num_rois, num_channels, pooled_height, pooled_width]
            const auto out_spatial_size = pooled_height * pooled_width;
            const auto out_roi_size = num_channels * out_spatial_size;

            /* we have to compute the output value for every combination of (roi, c, y, x) in the output
             *
             * the computation involving (y, x) are identical for all non-spatial dimensions
             * the computation and memory requests involving the roi are identical for remaining three axes
             *
             * we process multiple channels every iteration to reuse the identical computation
             * and memory requests involved with the roi and spatial dimensions
             */
            /*
             * if we are processing `CHANNELS_PER_ITER` channels per iteration, we will need
             * (num_channels / CHANNELS_PER_ITER) iterations per (roi, x, y)
             */
            auto num_channel_iters_per_roi_xy = num_channels / CHANNELS_PER_ITER;

            /* we need `num_channel_iters_per_roi_xy` iterations per (roi, x, y) and there are
             * `num_rois` rois and `out_spatial_size` combinations of (x, y)
             */
            auto iters_per_roi = num_channel_iters_per_roi_xy * out_spatial_size;
            auto iters_required = num_rois * iters_per_roi;

            for (auto iter : grid_stride_range(iters_required))
            {
                const index_type roi_no = iter / iters_per_roi;
                const index_type c_start = ((iter % iters_per_roi) / out_spatial_size) * CHANNELS_PER_ITER;

                /* note here that consecutive `iter` values will often have consecutive `x` values
                 * => stores into output will be coalesced across threads
                 */
                const index_type y = (iter % out_spatial_size) / pooled_width;
                const index_type x = iter % pooled_width;

                const index_type roi_offset = roi_no * 5;

                using device::round;
                const index_type batch_id = rois[roi_offset + 0];
                const index_type x_start_roi = round(static_cast<float>(rois[roi_offset + 1]) * spatial_scale);
                const index_type y_start_roi = round(static_cast<float>(rois[roi_offset + 2]) * spatial_scale);
                const index_type x_end_roi = round(static_cast<float>(rois[roi_offset + 3]) * spatial_scale);
                const index_type y_end_roi = round(static_cast<float>(rois[roi_offset + 4]) * spatial_scale);

                using device::max;
                const auto roi_width = max<index_type>(x_end_roi - x_start_roi + 1, 1);
                const auto roi_height = max<index_type>(y_end_roi - y_start_roi + 1, 1);

                const auto roi_width_ratio = static_cast<float>(roi_width) / pooled_width;
                const auto roi_height_ratio = static_cast<float>(roi_height) / pooled_height;

                auto x_start = x_start_roi + static_cast<index_type>(x * roi_width_ratio);
                auto y_start = y_start_roi + static_cast<index_type>(y * roi_height_ratio);

                using device::ceil;
                auto x_end = x_start_roi + static_cast<index_type>(ceil((x + 1) * roi_width_ratio));
                auto y_end = y_start_roi + static_cast<index_type>(ceil((y + 1) * roi_height_ratio));

                using device::max;
                x_start = max<index_type>(x_start, 0);
                y_start = max<index_type>(y_start, 0);

                using device::min;
                x_end = min<index_type>(x_end, in_width);
                y_end = min<index_type>(y_end, in_height);

                index_type in_offset = (batch_id * num_channels + c_start) * in_height * in_width;
                index_type out_idx = roi_no * out_roi_size + c_start * out_spatial_size + y * pooled_width + x;

                for (int i = 0; i < CHANNELS_PER_ITER; i++)
                {
                    /* We have to set the output to zero if (x_start >= x_end) or (y_start >= y_end). If either
                     * condition is true, the loops below won't execute even a single iteration. Hence, by setting
                     * `max_val` to zero in this case, we can combine it with the `else` code.
                     */
                    T max_val = (x_start >= x_end || y_start >= y_end) ? T(0) : device::numeric_limits<T>::lowest();

                    for (auto iy = y_start; iy < y_end; iy++)
                    {
                        const auto in_idx = in_offset + iy * in_width;
                        for (auto ix = x_start; ix < x_end; ix++)
                        {
                            max_val = max(max_val, load_ldg(input[in_idx + ix]));
                        }
                    }

                    output[out_idx] = max_val;

                    in_offset += in_image_size;
                    out_idx += out_spatial_size;
                }
            }
        }
    }

    template <class T, std::size_t CHANNELS_PER_ITER> static
    void launch_multichannel_roi_pooling(const Stream& stream,
        Span<T> output, size_type pooled_height, size_type pooled_width,
        View<T> input, size_type in_height, size_type in_width,
        View<T> rois, size_type num_channels, float spatial_scale)
    {
        auto kernel = raw::roi_pooling<T, CHANNELS_PER_ITER>;
        auto policy = make_policy(kernel, output.size() / CHANNELS_PER_ITER, 0, stream);
        launch_kernel(kernel, policy, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
    }

    template <class T>
    void roi_pooling(const Stream& stream, TensorSpan<T> output, TensorView<T> input, View<T> rois, float spatial_scale)
    {
        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        size_type num_channels = output.get_axis_size(1);

        size_type pooled_height = output.get_axis_size(2);
        size_type pooled_width = output.get_axis_size(3);

        size_type in_height = input.get_axis_size(2);
        size_type in_width = input.get_axis_size(3);

        if (num_channels % 64 == 0) {
            launch_multichannel_roi_pooling<T, 64>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        } else if (num_channels % 32 == 0) {
            launch_multichannel_roi_pooling<T, 32>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        } else if (num_channels % 16 == 0) {
            launch_multichannel_roi_pooling<T, 16>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        } else if (num_channels % 8 == 0) {
            launch_multichannel_roi_pooling<T, 8>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        } else if (num_channels % 4 == 0) {
            launch_multichannel_roi_pooling<T, 4>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        } else if (num_channels % 2 == 0) {
            launch_multichannel_roi_pooling<T, 2>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        } else {
            launch_multichannel_roi_pooling<T, 1>(stream, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void roi_pooling(const Stream& stream, TensorSpan<__half> output, TensorView<__half> input, View<__half> rois, float spatial_scale);
#endif
    template void roi_pooling(const Stream& stream, TensorSpan<float> output, TensorView<float> input, View<float> rois, float spatial_scale);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
