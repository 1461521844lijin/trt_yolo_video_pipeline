#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "functors.hpp"
#include "types.hpp"
#include "vector_traits.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

namespace raw {

    template <class T, class EltwiseOp, class ActivationOp, std::size_t N>
    __global__ void eltwise_op_generic_op_vec(Span<T> output, View<T> x, View<T> y, const typename EltwiseOp::Params eltwise_params, const typename ActivationOp::Params act_params) {
        using vector_type = get_vector_type_t<T, N>;

        auto output_vPtr = vector_type::get_pointer(output.data());
        auto x_vPtr = vector_type::get_pointer(x.data());
        auto y_vPtr = vector_type::get_pointer(y.data());

        EltwiseOp eltwise_op(eltwise_params);
        ActivationOp activation_op(act_params);

        for (auto i : grid_stride_range(output.size() / vector_type::size())) {
            vector_type vec_x, vec_y;
            v_load(vec_x, x_vPtr[i]);
            v_load(vec_y, y_vPtr[i]);
            for(int j = 0; j < vec_x.size(); j++)
                vec_x.data[j] = activation_op(eltwise_op(vec_x.data[j], vec_y.data[j]));
            v_store(output_vPtr[i], vec_x);
        }
    }
}

template <class T, class EltwiseOp, class ActivationOp, std::size_t N> static
void launch_vectorized_eltwise_op_generic_op(const Stream& stream, Span<T> output, View<T> x, View<T> y, const typename EltwiseOp::Params& eltwise_params, const typename ActivationOp::Params& act_params) {
    CV_Assert(is_fully_aligned<T>(output, N));
    CV_Assert(is_fully_aligned<T>(x, N));
    CV_Assert(is_fully_aligned<T>(y, N));

    auto kernel = raw::eltwise_op_generic_op_vec<T, EltwiseOp, ActivationOp, N>;
    auto policy = make_policy(kernel, output.size() / N, 0, stream);
    launch_kernel(kernel, policy, output, x, y, eltwise_params, act_params);
}

template <class T, class EltwiseOp, class ActivationOp> static
void eltwise_op_generic_op(const Stream& stream, Span<T> output, View<T> x, View<T> y, const typename EltwiseOp::Params& eltwise_params = {}, const typename ActivationOp::Params& act_params = {}) {
    CV_Assert(output.size() == x.size());
    CV_Assert(output.size() == y.size());

    if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
        launch_vectorized_eltwise_op_generic_op<T, EltwiseOp, ActivationOp, 4>(stream, output, x, y, eltwise_params, act_params);
    } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 4)) {
        launch_vectorized_eltwise_op_generic_op<T, EltwiseOp, ActivationOp, 2>(stream, output, x, y, eltwise_params, act_params);
    } else {
        launch_vectorized_eltwise_op_generic_op<T, EltwiseOp, ActivationOp, 1>(stream, output, x, y, eltwise_params, act_params);
    }
}

template <class T>
void eltwise_sum_2_relu(const Stream& stream, Span<T> output, View<T> x, View<T> y, T slope) {
    eltwise_op_generic_op<T, SumFunctor<T>, ReLUFunctor<T>>(stream, output, x, y, {}, {slope});
}

template <class T>
void eltwise_sum_2_clipped_relu(const Stream& stream, Span<T> output, View<T> x, View<T> y, T floor, T ceiling) {
    CV_Assert(static_cast<double>(floor) <= static_cast<double>(ceiling));
    eltwise_op_generic_op<T, SumFunctor<T>, ClippedReLUFunctor<T>>(stream, output, x, y, {}, {floor, ceiling});
}

template <class T>
void eltwise_sum_2_tanh(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op_generic_op<T, SumFunctor<T>, TanHFunctor<T>>(stream, output, x, y);
}

template <class T>
void eltwise_sum_2_swish(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op_generic_op<T, SumFunctor<T>, SwishFunctor<T>>(stream, output, x, y);
}

template <class T>
void eltwise_sum_2_mish(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op_generic_op<T, SumFunctor<T>, MishFunctor<T>>(stream, output, x, y);
}

template <class T>
void eltwise_sum_2_sigmoid(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op_generic_op<T, SumFunctor<T>, SigmoidFunctor<T>>(stream, output, x, y);
}

template <class T>
void eltwise_sum_2_power(const Stream& stream, Span<T> output, View<T> x, View<T> y, T exp, T scale, T shift) {
    eltwise_op_generic_op<T, SumFunctor<T>, PowerFunctor<T>>(stream, output, x, y, {}, {exp, scale, shift});
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
template void eltwise_sum_2_relu<__half>(const Stream&, Span<__half>, View<__half>, View<__half>, __half);
template void eltwise_sum_2_clipped_relu<__half>(const Stream&, Span<__half>, View<__half>, View<__half>, __half, __half);
template void eltwise_sum_2_tanh<__half>(const Stream&, Span<__half>, View<__half>, View<__half>);
template void eltwise_sum_2_swish<__half>(const Stream&, Span<__half>, View<__half>, View<__half>);
template void eltwise_sum_2_mish<__half>(const Stream&, Span<__half>, View<__half>, View<__half>);
template void eltwise_sum_2_sigmoid<__half>(const Stream&, Span<__half>, View<__half>, View<__half>);
template void eltwise_sum_2_power<__half>(const Stream&, Span<__half>, View<__half>, View<__half>, __half, __half, __half);
#endif

template void eltwise_sum_2_relu<float>(const Stream&, Span<float>, View<float>, View<float>, float);
template void eltwise_sum_2_clipped_relu<float>(const Stream&, Span<float>, View<float>, View<float>, float, float);
template void eltwise_sum_2_tanh<float>(const Stream&, Span<float>, View<float>, View<float>);
template void eltwise_sum_2_swish<float>(const Stream&, Span<float>, View<float>, View<float>);
template void eltwise_sum_2_mish<float>(const Stream&, Span<float>, View<float>, View<float>);
template void eltwise_sum_2_sigmoid<float>(const Stream&, Span<float>, View<float>, View<float>);
template void eltwise_sum_2_power<float>(const Stream&, Span<float>, View<float>, View<float>, float, float, float);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
