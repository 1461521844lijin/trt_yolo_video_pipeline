#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "array.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "kernel_dispatcher.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include "../cuda4dnn/kernels/fill_copy.hpp"

#include <opencv2/core.hpp>

#include <cstddef>
#include <vector>
#include <iostream>
#include <algorithm>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, std::size_t Rank>
        __global__ void slice(
            Span<T> output, array<size_type, Rank> out_strides,
            View<T> input, array<size_type, Rank> in_strides, array<index_type, Rank> in_offset)
        {
            for (auto i : grid_stride_range(output.size())) {
                index_type out_index = i / out_strides[0];
                index_type in_index = in_offset[0] + out_index;
                index_type iidx = in_index * in_strides[0];
                for (int j = 1; j < Rank; j++) {
                    out_index = (i % out_strides[j - 1]) / out_strides[j];
                    in_index = in_offset[j] + out_index;
                    iidx += in_index * in_strides[j];
                }

                output[i] = input[iidx];
            }
        }
    }

    template <class T, std::size_t Rank> static
    void launch_slice(
        const Stream& stream,
        Span<T> output, const std::vector<std::size_t>& outStride,
        View<T> input, const std::vector<std::size_t>& inStride, const std::vector<std::size_t>& inOffset)
    {
        CV_Assert(outStride.size() == Rank);
        CV_Assert(inStride.size() == Rank);
        CV_Assert(inOffset.size() == Rank);

        array<size_type, Rank> outStride_k, inStride_k;
        outStride_k.assign(std::begin(outStride), std::end(outStride));
        inStride_k.assign(std::begin(inStride), std::end(inStride));

        array<index_type, Rank> inOffset_k;
        inOffset_k.assign(std::begin(inOffset), std::end(inOffset));

        auto kernel = raw::slice<T, Rank>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, outStride_k, input, inStride_k, inOffset_k);
    }

    GENERATE_KERNEL_DISPATCHER(slice_dispatcher, launch_slice);

    template <class T>
    void slice(const Stream& stream,
        TensorSpan<T> output, TensorView<T> input,
        std::vector<std::size_t> offsets)
    {
        CV_Assert(output.rank() == input.rank());
        CV_Assert(output.rank() == offsets.size());

        /* copy directly if no slicing is required */
        if (is_shape_same(output, input))
        {
            CV_Assert(std::all_of(std::begin(offsets), std::end(offsets), [] (std::size_t x) { return x == 0; }));
            kernels::copy<T>(stream, output, input);
            return;
        }

        /* squeezable axes at the beginning of both tensors can be eliminated
         *
         * Reasoning:
         * ----------
         * Suppose an item's indices in the output tensor is [o1, o2, ...]. The indices in the input
         * tensor will be [o1 + off1, o2 + off2, ...]. The rest of the elements in the input are ignored.
         *
         * If the size of the first axis of the input and output tensor is unity, the input and output indices
         * for all the elements will be of the form be [0, o2 + off2, ...] and [0, o2, ...] respectively. Note that
         * there cannot be any ignored items since the axes have unit size. The first index does not contribute to the
         * element's address calculation and hence does nothing apart from eating up few cycles.
         */
        while (input.get_axis_size(0) == 1 && output.get_axis_size(0) == 1) {
            CV_Assert(offsets[0] == 0);

            input.squeeze(0);
            output.squeeze(0);
            offsets.erase(std::begin(offsets));

            CV_Assert(output.rank() == input.rank());
            CV_Assert(output.rank() == offsets.size());
        }

        auto inShape = input.shape_as_vector();
        auto outShape = output.shape_as_vector();

        /* contiguous axes which do not undergo slicing can be combined into one axis
         *
         * Reasoning:
         * ----------
         * Suppose an item's indices in the output tensor is [o1, o2, o3, ...]. Let the first two axes not undergo any
         * slicing. The indices in the input tensor will be [o1, o2, o3 + off3, ...].
         *
         * Each axis in the contiguous unsliced axes sequence will add an offset of iN * strideN. In the above example,
         * the two axes add a total offset of `o1 * stride1 + o2 * stride2`. We can merge the two axes into one axis with
         * a size of `size1 * size2`. The new offset added will be o12 * stride2` as the kernel iterates through `o12`.
         * Note that `o12` is actually `(o1 * size2 + o2)` in the original tensor.
         */
        for (int i = 0; i < inShape.size(); i++) {
            /* check if axis `i` requires any slicing */
            if (offsets[i] == 0 && inShape[i] == outShape[i]) {
                /* loop invariant: `i` is the first axis in the contiguous unsliced axis sequence */

                int j = i + 1; /* `j` is the axis which we will attempt to merge */
                while (j < inShape.size() && offsets[j] == 0 && inShape[j] == outShape[j]) {
                    /* `j` axis is also unsliced; merge `i` and `j` */
                    auto new_size = inShape[i] * inShape[j];
                    inShape[i] = new_size;
                    outShape[i] = new_size;
                    offsets[i] = 0; /* redundant */

                    /* delete axis `j` */
                    inShape.erase(std::begin(inShape) + j);
                    outShape.erase(std::begin(outShape) + j);
                    offsets.erase(std::begin(offsets) + j);

                    /* optimizations should not break the invariants */
                    CV_Assert(inShape.size() == outShape.size());
                    CV_Assert(inShape.size() == offsets.size());
                    CV_Assert(inShape[i] == outShape[i]);
                    CV_Assert(offsets[i] == 0);
                }
            }
        }

        auto rank = inShape.size();

        /* We can do a copy if the reduced rank is two and only the first axis is sliced.
         * The general requirement is that only one axis is sliced and all the axes that
         * precede the sliced axis are singleton. However, the reductions above will remove
         * all the leading singleton axes and merge the trailing unsliced axes into one, or
         * zero if there are no trailing unsliced axes. The latter is handled separately.
         */
        if (rank == 2 && offsets[0] != 0 && offsets[1] == 0)
        {
            auto stride = inShape[1];
            auto sliced_input = View<T>(input.get() + offsets[0] * stride, output.size());
            kernels::copy<T>(stream, output, sliced_input);
            return;
        }

        if (rank == 1)
        {
            auto sliced_input = View<T>(input.get() + offsets[0], output.size());
            kernels::copy<T>(stream, output, sliced_input);
            return;
        }

        std::vector<std::size_t> inStride(rank), outStride(rank);
        inStride.back() = 1;
        outStride.back() = 1;
        /* garbage, ..., garbage, 1 */

        std::copy(std::begin(inShape) + 1, std::end(inShape), std::begin(inStride));
        std::copy(std::begin(outShape) + 1, std::end(outShape), std::begin(outStride));
        /* dim[0], dim[1], ..., dim[-1], 1 */

        std::partial_sum(inStride.rbegin(), inStride.rend(), inStride.rbegin(), std::multiplies<std::size_t>());
        std::partial_sum(outStride.rbegin(), outStride.rend(), outStride.rbegin(), std::multiplies<std::size_t>());
        /* stride[0], stride[1], ..., stride[-2], 1 */

        CV_Assert(1 <= rank && rank <= CSL_MAX_TENSOR_RANK);
        slice_dispatcher<T, 1, CSL_MAX_TENSOR_RANK>(rank, stream, output, outStride, input, inStride, offsets);
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void slice(const Stream&, TensorSpan<__half>, TensorView<__half>, std::vector<std::size_t>);
#endif
    template void slice(const Stream&, TensorSpan<float>, TensorView<float>, std::vector<std::size_t>);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
