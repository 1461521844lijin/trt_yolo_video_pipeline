#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "array.hpp"
#include "math.hpp"
#include "types.hpp"
#include "vector_traits.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <cstddef>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, bool Normalize>
        __global__ void prior_box(
            Span<T> output,
            View<float> boxWidth, View<float> boxHeight, View<float> offsetX, View<float> offsetY, float stepX, float stepY,
            size_type layerWidth, size_type layerHeight,
            size_type imageWidth, size_type imageHeight)
        {
            /* each box consists of two pair of coordinates and hence 4 values in total */
            /* since the entire output consists (first channel at least) of these boxes,
             * we are garunteeed that the output is aligned to a boundary of 4 values
             */
            using vector_type = get_vector_type_t<T, 4>;
            auto output_vPtr = vector_type::get_pointer(output.data());

            /* num_points contains the number of points in the feature map of interest
             * each iteration of the stride loop selects a point and generates prior boxes for it
             */
            size_type num_points = layerWidth * layerHeight;
            for (auto idx : grid_stride_range(num_points)) {
                const index_type x = idx % layerWidth,
                                 y = idx / layerWidth;

                index_type output_offset_v4 = idx * offsetX.size() * boxWidth.size();
                for (int i = 0; i < boxWidth.size(); i++) {
                    for (int j = 0; j < offsetX.size(); j++) {
                        float center_x = (x + offsetX[j]) * stepX;
                        float center_y = (y + offsetY[j]) * stepY;

                        vector_type vec;
                        if(Normalize) {
                            vec.data[0] = (center_x - boxWidth[i] * 0.5f) / imageWidth;
                            vec.data[1] = (center_y - boxHeight[i] * 0.5f) / imageHeight;
                            vec.data[2] = (center_x + boxWidth[i] * 0.5f) / imageWidth;
                            vec.data[3] = (center_y + boxHeight[i] * 0.5f) / imageHeight;
                        } else {
                            vec.data[0] = center_x - boxWidth[i] * 0.5f;
                            vec.data[1] = center_y - boxHeight[i] * 0.5f;
                            vec.data[2] = center_x + boxWidth[i] * 0.5f - 1.0f;
                            vec.data[3] = center_y + boxHeight[i] * 0.5f - 1.0f;
                        }

                        v_store(output_vPtr[output_offset_v4], vec);
                        output_offset_v4++;
                    }
                }
            }
        }

        template <class T>
        __global__ void prior_box_clip(Span<T> output) {
            for (auto i : grid_stride_range(output.size())) {
                using device::clamp;
                output[i] = clamp<T>(output[i], 0.0, 1.0);
            }
        }

        template <class T>
        __global__ void prior_box_set_variance1(Span<T> output, float variance) {
            using vector_type = get_vector_type_t<T, 4>;
            auto output_vPtr = vector_type::get_pointer(output.data());
            for (auto i : grid_stride_range(output.size() / 4)) {
                vector_type vec;
                for (int j = 0; j < 4; j++)
                    vec.data[j] = variance;
                v_store(output_vPtr[i], vec);
            }
        }

        template <class T>
        __global__ void prior_box_set_variance4(Span<T> output, array<float, 4> variance) {
            using vector_type = get_vector_type_t<T, 4>;
            auto output_vPtr = vector_type::get_pointer(output.data());
            for (auto i : grid_stride_range(output.size() / 4)) {
                vector_type vec;
                for(int j = 0; j < 4; j++)
                    vec.data[j] = variance[j];
                v_store(output_vPtr[i], vec);
            }
        }
    }

    template <class T, bool Normalize> static
    void launch_prior_box_kernel(
        const Stream& stream,
        Span<T> output, View<float> boxWidth, View<float> boxHeight, View<float> offsetX, View<float> offsetY, float stepX, float stepY,
        std::size_t layerWidth, std::size_t layerHeight, std::size_t imageWidth, std::size_t imageHeight)
    {
        auto num_points = layerWidth * layerHeight;
        auto kernel = raw::prior_box<T, Normalize>;
        auto policy = make_policy(kernel, num_points, 0, stream);
        launch_kernel(kernel, policy,
            output, boxWidth, boxHeight, offsetX, offsetY, stepX, stepY,
            layerWidth, layerHeight, imageWidth, imageHeight);
    }

    template <class T>
    void generate_prior_boxes(
        const Stream& stream,
        Span<T> output,
        View<float> boxWidth, View<float> boxHeight, View<float> offsetX, View<float> offsetY, float stepX, float stepY,
        std::vector<float> variance,
        std::size_t numPriors,
        std::size_t layerWidth, std::size_t layerHeight,
        std::size_t imageWidth, std::size_t imageHeight,
        bool normalize, bool clip)
    {
        if (normalize) {
            launch_prior_box_kernel<T, true>(
                stream, output, boxWidth, boxHeight, offsetX, offsetY, stepX, stepY,
                layerWidth, layerHeight, imageWidth, imageHeight
            );
        } else {
            launch_prior_box_kernel<T, false>(
                stream, output, boxWidth, boxHeight, offsetX, offsetY, stepX, stepY,
                layerWidth, layerHeight, imageWidth, imageHeight
            );
        }

        std::size_t channel_size = layerHeight * layerWidth * numPriors * 4;
        CV_Assert(channel_size * 2 == output.size());

        if (clip) {
            auto output_span_c1 = Span<T>(output.data(), channel_size);
            auto kernel = raw::prior_box_clip<T>;
            auto policy = make_policy(kernel, output_span_c1.size(), 0, stream);
            launch_kernel(kernel, policy, output_span_c1);
        }

        auto output_span_c2 = Span<T>(output.data() + channel_size, channel_size);
        if (variance.size() == 1) {
            auto kernel = raw::prior_box_set_variance1<T>;
            auto policy = make_policy(kernel, output_span_c2.size() / 4, 0, stream);
            launch_kernel(kernel, policy, output_span_c2, variance[0]);
        } else {
            array<float, 4> variance_k;
            variance_k.assign(std::begin(variance), std::end(variance));
            auto kernel = raw::prior_box_set_variance4<T>;
            auto policy = make_policy(kernel, output_span_c2.size() / 4, 0, stream);
            launch_kernel(kernel, policy, output_span_c2, variance_k);
        }
    }

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void generate_prior_boxes(const Stream&, Span<__half>, View<float>, View<float>, View<float>, View<float>, float, float,
        std::vector<float>, std::size_t, std::size_t, std::size_t, std::size_t, std::size_t, bool, bool);
#endif

    template void generate_prior_boxes(const Stream&, Span<float>, View<float>, View<float>, View<float>, View<float>, float, float,
        std::vector<float>, std::size_t, std::size_t, std::size_t, std::size_t, std::size_t, bool, bool);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
