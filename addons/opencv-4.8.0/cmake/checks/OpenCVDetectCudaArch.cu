
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <list>

int main()
{
    std::ostringstream arch;
    std::list<std::string> archs;

    int count = 0;
    if (hipSuccess != hipGetDeviceCount(&count)){ return -1; }
    if (count == 0) { return -1; }
    for (int device = 0; device < count; ++device)
    {
        hipDeviceProp_t prop;
        if (hipSuccess != hipGetDeviceProperties(&prop, device)){ continue; }
        arch << prop.major << "." << prop.minor;
        archs.push_back(arch.str());
        arch.str("");
    }
    archs.unique(); // Some devices might have the same arch
    for (std::list<std::string>::iterator it=archs.begin(); it!=archs.end(); ++it)
        std::cout << *it << " ";
    return 0;
}
