#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/filters.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        struct AffineTransform
        {
            static const int rows = 2;
            static __device__ __forceinline__ float2 calcCoord(const float warpMat[AffineTransform::rows * 3], int x, int y)
            {
                const float xcoo = warpMat[0] * x + warpMat[1] * y + warpMat[2];
                const float ycoo = warpMat[3] * x + warpMat[4] * y + warpMat[5];

                return make_float2(xcoo, ycoo);
            }

            struct Coefficients
            {
                Coefficients(const float* c_)
                {
                    for(int i = 0; i < AffineTransform::rows * 3; i++)
                        c[i] = c_[i];
                }
                float c[AffineTransform::rows * 3];
            };
        };

        struct PerspectiveTransform
        {
            static const int rows = 3;
            static __device__ __forceinline__ float2 calcCoord(const float warpMat[PerspectiveTransform::rows * 3], int x, int y)
            {
                const float coeff = 1.0f / (warpMat[6] * x + warpMat[7] * y + warpMat[8]);

                const float xcoo = coeff * (warpMat[0] * x + warpMat[1] * y + warpMat[2]);
                const float ycoo = coeff * (warpMat[3] * x + warpMat[4] * y + warpMat[5]);

                return make_float2(xcoo, ycoo);
            }
            struct Coefficients
            {
                Coefficients(const float* c_)
                {
                    for(int i = 0; i < PerspectiveTransform::rows * 3; i++)
                        c[i] = c_[i];
                }

                float c[PerspectiveTransform::rows * 3];
            };
        };

        ///////////////////////////////////////////////////////////////////
        // Build Maps

        template <class Transform> __global__ void buildWarpMaps(PtrStepSzf xmap, PtrStepf ymap, const typename Transform::Coefficients warpMat)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < xmap.cols && y < xmap.rows)
            {
                const float2 coord = Transform::calcCoord(warpMat.c, x, y);

                xmap(y, x) = coord.x;
                ymap(y, x) = coord.y;
            }
        }

        template <class Transform> void buildWarpMaps_caller(PtrStepSzf xmap, PtrStepSzf ymap, const float warpMat[Transform::rows * 3], hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(xmap.cols, block.x), divUp(xmap.rows, block.y));

            buildWarpMaps<Transform><<<grid, block, 0, stream>>>(xmap, ymap, warpMat);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void buildWarpAffineMaps_gpu(float coeffs[2 * 3], PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            buildWarpMaps_caller<AffineTransform>(xmap, ymap, coeffs, stream);
        }

        void buildWarpPerspectiveMaps_gpu(float coeffs[3 * 3], PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            buildWarpMaps_caller<PerspectiveTransform>(xmap, ymap, coeffs, stream);
        }

        ///////////////////////////////////////////////////////////////////
        // Warp

        template <class Transform, class Ptr2D, typename T> __global__ void warp(const Ptr2D src, PtrStepSz<T> dst, const typename Transform::Coefficients warpMat)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                const float2 coord = Transform::calcCoord(warpMat.c, x, y);

                dst.ptr(y)[x] = saturate_cast<T>(src(coord.y, coord.x));
            }
        }

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool)
            {
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader<PtrStep<T>, B<work_type>> brdSrc(src, brd);
                Filter<BorderReader<PtrStep<T>, B<work_type>>> filter_src(brdSrc);

                warp<Transform><<<grid, block, 0, stream>>>(filter_src, dst, warpMat);
                cudaSafeCall( hipGetLastError() );
            }
        };

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherNonStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], bool)
            {
                CV_UNUSED(xoff);
                CV_UNUSED(yoff);
                CV_UNUSED(srcWhole);

                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader<PtrStep<T>, B<work_type>> brdSrc(src, brd);
                Filter<BorderReader<PtrStep<T>, B<work_type>>> filter_src(brdSrc);

                warp<Transform><<<grid, block>>>(filter_src, dst, warpMat);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherNonStreamTex
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], bool cc20)
            {
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;
                dim3 block(32, cc20 ? 8 : 4);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));
                if (xoff || yoff) {
                    cudev::TextureOff<T> texSrcWhole(srcWhole, yoff, xoff);
                    B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                    BorderReader<cudev::TextureOffPtr<T>, B<work_type>> brdSrc(texSrcWhole, brd);
                    Filter<BorderReader<cudev::TextureOffPtr<T>, B<work_type>>> filter_src(brdSrc);
                    warp<Transform><<<grid, block>>> (filter_src, dst, warpMat);
                }
                else {
                    cudev::Texture<T> texSrcWhole(srcWhole);
                    B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                    BorderReader<cudev::TexturePtr<T>, B<work_type>>brdSrc(texSrcWhole, brd);
                    Filter< BorderReader<cudev::TexturePtr<T>, B<work_type>>> filter_src(brdSrc);
                    warp<Transform><<<grid, block>>> (filter_src, dst, warpMat);
                }
                cudaSafeCall( hipGetLastError() );
                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, uchar> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, uchar> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, uchar4> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, uchar4> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, ushort> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, ushort> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, ushort4> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, ushort4> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, short> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, short> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, short4> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, short4> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, float> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, float> {};
        template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, float4> :
            WarpDispatcherNonStreamTex<Transform, Filter, B, float4> {};

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcher
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool cc20)
            {
                if (stream == 0)
                    WarpDispatcherNonStream<Transform, Filter, B, T>::call(src, srcWhole, xoff, yoff, dst, borderValue, warpMat, cc20);
                else
                    WarpDispatcherStream<Transform, Filter, B, T>::call(src, dst, borderValue, warpMat, stream, cc20);
            }
        };

        template <class Transform, typename T>
        void warp_caller(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzb dst, int interpolation,
                         int borderMode, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool cc20)
        {
            typedef void (*func_t)(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool cc20);

            static const func_t funcs[3][5] =
            {
                {
                    WarpDispatcher<Transform, PointFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReflect101, T>::call
                },
                {
                    WarpDispatcher<Transform, LinearFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReflect101, T>::call
                },
                {
                    WarpDispatcher<Transform, CubicFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReflect101, T>::call
                }
            };

            funcs[interpolation][borderMode](static_cast<PtrStepSz<T>>(src), static_cast<PtrStepSz<T>>(srcWhole), xoff, yoff,
                                             static_cast<PtrStepSz<T>>(dst), borderValue, warpMat, stream, cc20);
        }

        template <typename T> void warpAffine_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation,
                                                  int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            warp_caller<AffineTransform, T>(src, srcWhole, xoff, yoff, dst, interpolation, borderMode, borderValue, coeffs, stream, cc20);
        }

        template void warpAffine_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template <typename T> void warpPerspective_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation,
                                                  int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            warp_caller<PerspectiveTransform, T>(src, srcWhole, xoff, yoff, dst, interpolation, borderMode, borderValue, coeffs, stream, cc20);
        }

        template void warpPerspective_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
