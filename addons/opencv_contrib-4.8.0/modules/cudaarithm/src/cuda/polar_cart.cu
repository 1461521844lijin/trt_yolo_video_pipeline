#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

void cv::cuda::magnitude(InputArray _x, InputArray _y, OutputArray _dst, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat dst = getOutputMat(_dst, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(dst.reshape(1));

    gridTransformBinary(xc, yc, magc, magnitude_func<float>(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::magnitudeSqr(InputArray _x, InputArray _y, OutputArray _dst, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat dst = getOutputMat(_dst, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(dst.reshape(1));

    gridTransformBinary(xc, yc, magc, magnitude_sqr_func<float>(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::phase(InputArray _x, InputArray _y, OutputArray _dst, bool angleInDegrees, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat dst = getOutputMat(_dst, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> anglec(dst.reshape(1));

    if (angleInDegrees)
        gridTransformBinary(xc, yc, anglec, direction_func<float, true>(), stream);
    else
        gridTransformBinary(xc, yc, anglec, direction_func<float, false>(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::cartToPolar(InputArray _x, InputArray _y, OutputArray _mag, OutputArray _angle, bool angleInDegrees, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat mag = getOutputMat(_mag, x.size(), CV_32FC1, stream);
    GpuMat angle = getOutputMat(_angle, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(mag.reshape(1));
    GpuMat_<float> anglec(angle.reshape(1));

    if (angleInDegrees)
    {
        gridTransformTuple(zipPtr(xc, yc),
                           tie(magc, anglec),
                           make_tuple(
                               binaryTupleAdapter<0, 1>(magnitude_func<float>()),
                               binaryTupleAdapter<0, 1>(direction_func<float, true>())),
                           stream);
    }
    else
    {
        gridTransformTuple(zipPtr(xc, yc),
                           tie(magc, anglec),
                           make_tuple(
                               binaryTupleAdapter<0, 1>(magnitude_func<float>()),
                               binaryTupleAdapter<0, 1>(direction_func<float, false>())),
                           stream);
    }

    syncOutput(mag, _mag, stream);
    syncOutput(angle, _angle, stream);
}

namespace
{
    template <typename T> struct sincos_op
    {
        __device__ __forceinline__ void operator()(T a, T *sptr, T *cptr) const
        {
            ::sincos(a, sptr, cptr);
        }
    };
    template <> struct sincos_op<float>
    {
        __device__ __forceinline__ void operator()(float a, float *sptr, float *cptr) const
        {
            ::sincosf(a, sptr, cptr);
        }
    };

    template <typename T, bool useMag>
    __global__ void polarToCartImpl_(const GlobPtr<T> mag, const GlobPtr<T> angle, GlobPtr<T> xmat, GlobPtr<T> ymat, const T scale, const int rows, const int cols)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= cols || y >= rows)
            return;

        const T mag_val = useMag ? mag(y, x) : static_cast<T>(1.0);
        const T angle_val = angle(y, x);

        T sin_a, cos_a;
        sincos_op<T> op;
        op(scale * angle_val, &sin_a, &cos_a);

        xmat(y, x) = mag_val * cos_a;
        ymat(y, x) = mag_val * sin_a;
    }

    template <typename T>
    void polarToCartImpl(const GpuMat& mag, const GpuMat& angle, GpuMat& x, GpuMat& y, bool angleInDegrees, hipStream_t& stream)
    {
        GpuMat_<T> xc(x.reshape(1));
        GpuMat_<T> yc(y.reshape(1));
        GpuMat_<T> magc(mag.reshape(1));
        GpuMat_<T> anglec(angle.reshape(1));

        const dim3 block(32, 8);
        const dim3 grid(divUp(anglec.cols, block.x), divUp(anglec.rows, block.y));

        const T scale = angleInDegrees ? static_cast<T>(CV_PI / 180.0) : static_cast<T>(1.0);

        if (magc.empty())
            polarToCartImpl_<T, false> << <grid, block, 0, stream >> >(shrinkPtr(magc), shrinkPtr(anglec), shrinkPtr(xc), shrinkPtr(yc), scale, anglec.rows, anglec.cols);
        else
            polarToCartImpl_<T, true> << <grid, block, 0, stream >> >(shrinkPtr(magc), shrinkPtr(anglec), shrinkPtr(xc), shrinkPtr(yc), scale, anglec.rows, anglec.cols);
    }
}

void cv::cuda::polarToCart(InputArray _mag, InputArray _angle, OutputArray _x, OutputArray _y, bool angleInDegrees, Stream& _stream)
{
    typedef void(*func_t)(const GpuMat& mag, const GpuMat& angle, GpuMat& x, GpuMat& y, bool angleInDegrees, hipStream_t& stream);
    static const func_t funcs[7] = { 0, 0, 0, 0, 0, polarToCartImpl<float>, polarToCartImpl<double> };

    GpuMat mag = getInputMat(_mag, _stream);
    GpuMat angle = getInputMat(_angle, _stream);

    CV_Assert(angle.depth() == CV_32F || angle.depth() == CV_64F);
    CV_Assert( mag.empty() || (mag.type() == angle.type() && mag.size() == angle.size()) );

    GpuMat x = getOutputMat(_x, angle.size(), CV_MAKETYPE(angle.depth(), 1), _stream);
    GpuMat y = getOutputMat(_y, angle.size(), CV_MAKETYPE(angle.depth(), 1), _stream);

    hipStream_t stream = StreamAccessor::getStream(_stream);
    funcs[angle.depth()](mag, angle, x, y, angleInDegrees, stream);
    CV_CUDEV_SAFE_CALL( hipGetLastError() );

    syncOutput(x, _x, _stream);
    syncOutput(y, _y, _stream);

    if (stream == 0)
        CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

#endif
