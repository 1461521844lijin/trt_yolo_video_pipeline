#include "hip/hip_runtime.h"

#include "test_precomp.hpp"

using namespace cv;
using namespace cv::cudev;
using namespace cvtest;

// BlockScanInt

template <int THREADS_NUM>
__global__ void int_kernel(int* data)
{
    uint tid = Block::threadLineId();

#if CV_CUDEV_ARCH >= 300
    const int n_warps = (THREADS_NUM - 1) / WARP_SIZE + 1;
    __shared__ int smem[n_warps];
#else
    __shared__ int smem[THREADS_NUM];
#endif

    data[tid] = blockScanInclusive<THREADS_NUM>(data[tid], smem, tid);
}

#define BLOCK_SCAN_INT_TEST(block_size)                                 \
    TEST(BlockScanInt, BlockSize##block_size)                           \
    {                                                                   \
        Mat src = randomMat(Size(block_size, 1), CV_32SC1, 0, 1024);    \
                                                                        \
        GpuMat d_src;                                                   \
        d_src.upload(src);                                              \
                                                                        \
        for (int col = 1; col < block_size; col++)                      \
            src.at<int>(0, col) += src.at<int>(0, col - 1);             \
                                                                        \
        int_kernel<block_size><<<1, block_size>>>((int*)d_src.data);    \
                                                                        \
        CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());                    \
                                                                        \
        EXPECT_MAT_NEAR(d_src, src, 0);                                 \
    }

BLOCK_SCAN_INT_TEST(29)
BLOCK_SCAN_INT_TEST(30)
BLOCK_SCAN_INT_TEST(32)
BLOCK_SCAN_INT_TEST(40)
BLOCK_SCAN_INT_TEST(41)

BLOCK_SCAN_INT_TEST(59)
BLOCK_SCAN_INT_TEST(60)
BLOCK_SCAN_INT_TEST(64)
BLOCK_SCAN_INT_TEST(70)
BLOCK_SCAN_INT_TEST(71)

BLOCK_SCAN_INT_TEST(109)
BLOCK_SCAN_INT_TEST(110)
BLOCK_SCAN_INT_TEST(128)
BLOCK_SCAN_INT_TEST(130)
BLOCK_SCAN_INT_TEST(131)

BLOCK_SCAN_INT_TEST(189)
BLOCK_SCAN_INT_TEST(200)
BLOCK_SCAN_INT_TEST(256)
BLOCK_SCAN_INT_TEST(300)
BLOCK_SCAN_INT_TEST(311)

BLOCK_SCAN_INT_TEST(489)
BLOCK_SCAN_INT_TEST(500)
BLOCK_SCAN_INT_TEST(512)
BLOCK_SCAN_INT_TEST(600)
BLOCK_SCAN_INT_TEST(611)

BLOCK_SCAN_INT_TEST(1024)

// BlockScanDouble

template <int THREADS_NUM>
__global__ void double_kernel(double* data)
{
    uint tid = Block::threadLineId();

#if CV_CUDEV_ARCH >= 300
    const int n_warps = (THREADS_NUM - 1) / WARP_SIZE + 1;
    __shared__ double smem[n_warps];
#else
    __shared__ double smem[THREADS_NUM];
#endif

    data[tid] = blockScanInclusive<THREADS_NUM>(data[tid], smem, tid);
}

#define BLOCK_SCAN_DOUBLE_TEST(block_size)                                  \
    TEST(BlockScanDouble, BlockSize##block_size)                            \
    {                                                                       \
        Mat src = randomMat(Size(block_size, 1), CV_64FC1, 0.0, 1.0);       \
                                                                            \
        GpuMat d_src;                                                       \
        d_src.upload(src);                                                  \
                                                                            \
        for (int col = 1; col < block_size; col++)                          \
            src.at<double>(0, col) += src.at<double>(0, col - 1);           \
                                                                            \
        double_kernel<block_size><<<1, block_size>>>((double*)d_src.data);  \
                                                                            \
        CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());                        \
                                                                            \
        EXPECT_MAT_NEAR(d_src, src, 1e-10);                                 \
    }

BLOCK_SCAN_DOUBLE_TEST(29)
BLOCK_SCAN_DOUBLE_TEST(30)
BLOCK_SCAN_DOUBLE_TEST(32)
BLOCK_SCAN_DOUBLE_TEST(40)
BLOCK_SCAN_DOUBLE_TEST(41)

BLOCK_SCAN_DOUBLE_TEST(59)
BLOCK_SCAN_DOUBLE_TEST(60)
BLOCK_SCAN_DOUBLE_TEST(64)
BLOCK_SCAN_DOUBLE_TEST(70)
BLOCK_SCAN_DOUBLE_TEST(71)

BLOCK_SCAN_DOUBLE_TEST(109)
BLOCK_SCAN_DOUBLE_TEST(110)
BLOCK_SCAN_DOUBLE_TEST(128)
BLOCK_SCAN_DOUBLE_TEST(130)
BLOCK_SCAN_DOUBLE_TEST(131)

BLOCK_SCAN_DOUBLE_TEST(189)
BLOCK_SCAN_DOUBLE_TEST(200)
BLOCK_SCAN_DOUBLE_TEST(256)
BLOCK_SCAN_DOUBLE_TEST(300)
BLOCK_SCAN_DOUBLE_TEST(311)

BLOCK_SCAN_DOUBLE_TEST(489)
BLOCK_SCAN_DOUBLE_TEST(500)
BLOCK_SCAN_DOUBLE_TEST(512)
BLOCK_SCAN_DOUBLE_TEST(600)
BLOCK_SCAN_DOUBLE_TEST(611)

BLOCK_SCAN_DOUBLE_TEST(1024)
