#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "opencv2/core/cuda/utility.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>
#include <thrust/execution_policy.h>
namespace cv { namespace cuda { namespace device
{
    namespace gfft
    {
        template <class Mask> __global__ void findCorners(cv::cudev::TexturePtr<float> tex, float threshold, const Mask mask, float2* corners, int max_count, int rows, int cols, int *g_counter)
        {
            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (i > 0 && i < rows - 1 && j > 0 && j < cols - 1 && mask(i, j))
            {
                float val = tex(i, j);

                if (val > threshold)
                {
                    float maxVal = val;

                    maxVal = ::fmax(tex(i - 1, j - 1), maxVal);
                    maxVal = ::fmax(tex(i - 1, j), maxVal);
                    maxVal = ::fmax(tex(i - 1, j + 1), maxVal);

                    maxVal = ::fmax(tex(i, j - 1), maxVal);
                    maxVal = ::fmax(tex(i, j + 1), maxVal);

                    maxVal = ::fmax(tex(i + 1, j - 1), maxVal);
                    maxVal = ::fmax(tex(i + 1, j), maxVal);
                    maxVal = ::fmax(tex(i + 1, j + 1), maxVal);

                    if (val == maxVal)
                    {
                        const int ind = ::atomicAdd(g_counter, 1);

                        if (ind < max_count)
                            corners[ind] = make_float2(j, i);
                    }
                }
            }
        }

        int findCorners_gpu(const PtrStepSzf eig, float threshold, PtrStepSzb mask, float2* corners, int max_count, int* counterPtr, hipStream_t stream)
        {
            cudaSafeCall( hipMemsetAsync(counterPtr, 0, sizeof(int), stream) );
            cv::cudev::Texture<float> tex(eig);

            dim3 block(16, 16);
            dim3 grid(divUp(eig.cols, block.x), divUp(eig.rows, block.y));

            if (mask.data)
                findCorners<<<grid, block, 0, stream>>>(tex, threshold, SingleMask(mask), corners, max_count, eig.rows, eig.cols, counterPtr);
            else
                findCorners<<<grid, block, 0, stream>>>(tex, threshold, WithOutMask(), corners, max_count, eig.rows, eig.cols, counterPtr);

            cudaSafeCall( hipGetLastError() );

            int count;
            cudaSafeCall( hipMemcpyAsync(&count, counterPtr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            if (stream)
                cudaSafeCall(hipStreamSynchronize(stream));
            else
                cudaSafeCall( hipDeviceSynchronize() );
            return std::min(count, max_count);
        }

        class EigGreater
        {
        public:
            EigGreater(cv::cudev::TexturePtr<float> tex_) : tex(tex_) {}
            __device__ __forceinline__ bool operator()(float2 a, float2 b) const{
                return tex(a.y, a.x) > tex(b.y, b.x);
            }
            cv::cudev::TexturePtr<float> tex;
        };

        void sortCorners_gpu(const PtrStepSzf eig, float2* corners, int count, hipStream_t stream)
        {
            cv::cudev::Texture<float> tex(eig);
            thrust::device_ptr<float2> ptr(corners);
#if THRUST_VERSION >= 100802
            if (stream)
                thrust::sort(thrust::cuda::par(ThrustAllocator::getAllocator()).on(stream), ptr, ptr + count, EigGreater(tex));
            else
                thrust::sort(thrust::cuda::par(ThrustAllocator::getAllocator()), ptr, ptr + count, EigGreater(tex));
#else
            thrust::sort(ptr, ptr + count, EigGreater(tex));
#endif
        }
    } // namespace optical_flow
}}}


#endif /* CUDA_DISABLER */
