#include "hip/hip_runtime.h"
//
// Created by lijin on 2023/12/21.
//

#include "PostProcess.cuh"

#include "cuda_kernels/cuda_tools/cuda_tools.h"
#include <hip/hip_runtime.h>
#include <iostream>

namespace CUDA {

const int NUM_BOX_ELEMENT =
    8;  // left, top, right, bottom, confidence, class, keepflag, row_index(output)
static __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy) {
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel_common(float *predict,
                                            int    num_bboxes,
                                            int    num_classes,
                                            int    output_cdim,
                                            float  confidence_threshold,
                                            float *invert_affine_matrix,
                                            float *parray,
                                            int    MAX_IMAGE_BOXES) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes)
        return;

    float *pitem      = predict + output_cdim * position;
    float  objectness = pitem[4];
    if (objectness < confidence_threshold)
        return;

    float *class_confidence = pitem + 5;
    float  confidence       = *class_confidence++;
    int    label            = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label      = i;
        }
    }

    confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= MAX_IMAGE_BOXES)
        return;

    float cx     = *pitem++;
    float cy     = *pitem++;
    float width  = *pitem++;
    float height = *pitem++;
    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++     = left;
    *pout_item++     = top;
    *pout_item++     = right;
    *pout_item++     = bottom;
    *pout_item++     = confidence;
    *pout_item++     = label;
    *pout_item++     = 1;  // 1 = keep, 0 = ignore
}

static __global__ void yolov5_decode_kernel(float *predict,
                                            int    num_bboxes,
                                            int    num_classes,
                                            float  confidence_threshold,
                                            float *invert_affine_matrix,
                                            float *parray,
                                            int    max_objects) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes)
        return;

    float *pitem      = predict + (5 + num_classes) * position;
    float  objectness = pitem[4];
    if (objectness < confidence_threshold)
        return;

    float *class_confidence = pitem + 5;
    float  confidence       = *class_confidence++;
    int    label            = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label      = i;
        }
    }

    confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= max_objects)
        return;

    float cx     = *pitem++;
    float cy     = *pitem++;
    float width  = *pitem++;
    float height = *pitem++;
    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++     = left;
    *pout_item++     = top;
    *pout_item++     = right;
    *pout_item++     = bottom;
    *pout_item++     = confidence;
    *pout_item++     = label;
    *pout_item++     = 1;  // 1 = keep, 0 = ignore
}

static __device__ float box_iou(float aleft,
                                float atop,
                                float aright,
                                float abottom,
                                float bleft,
                                float btop,
                                float bright,
                                float bbottom) {
    float cleft   = max(aleft, bleft);
    float ctop    = max(atop, btop);
    float cright  = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void nms_kernel(float *bboxes, int max_objects, float threshold) {
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count    = min((int)*bboxes, max_objects);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) {
        float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5])
            continue;

        if (pitem[4] >= pcurrent[4]) {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0],
                                pitem[1], pitem[2], pitem[3]);

            if (iou > threshold) {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

void decode_kernel_common_invoker(float       *predict,
                                  int          num_bboxes,
                                  int          num_classes,
                                  int          output_cdim,
                                  float        confidence_threshold,
                                  float       *invert_affine_matrix,
                                  float       *parray,
                                  int          MAX_IMAGE_BOXES,
                                  hipStream_t stream) {
    auto grid  = CUDATools::grid_dims(num_bboxes);
    auto block = CUDATools::block_dims(num_bboxes);
    checkCudaKernel(decode_kernel_common<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
}

void yolov5_decode_kernel_invoker(float       *predict,
                                  int          num_bboxes,
                                  int          num_classes,
                                  float        confidence_threshold,
                                  float       *invert_affine_matrix,
                                  float       *parray,
                                  int          max_objects,
                                  hipStream_t stream) {
    auto grid  = CUDATools::grid_dims(num_bboxes);
    auto block = CUDATools::block_dims(num_bboxes);
    checkCudaKernel(yolov5_decode_kernel<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, confidence_threshold, invert_affine_matrix, parray,
        max_objects));
}

void nms_kernel_invoker(float *parray, float nms_threshold, int max_objects, hipStream_t stream) {
    auto grid  = CUDATools::grid_dims(max_objects);
    auto block = CUDATools::block_dims(max_objects);
    checkCudaKernel(nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold));
}

/////////////////////////////////////////////////// yolov8

static __global__ void decode_kernel_v8_trans(float *predict,
                                              int    num_bboxes,
                                              int    num_classes,
                                              int    output_cdim,
                                              float  confidence_threshold,
                                              float *invert_affine_matrix,
                                              float *parray,
                                              int    MAX_IMAGE_BOXES) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes)
        return;

    float *pitem            = predict + output_cdim * position;
    float *class_confidence = pitem + 4;
    float  confidence       = *class_confidence++;
    int    label            = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label      = i;
        }
    }
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= MAX_IMAGE_BOXES)
        return;

    float cx     = *pitem++;
    float cy     = *pitem++;
    float width  = *pitem++;
    float height = *pitem++;
    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++     = left;
    *pout_item++     = top;
    *pout_item++     = right;
    *pout_item++     = bottom;
    *pout_item++     = confidence;
    *pout_item++     = label;
    *pout_item++     = 1;  // 1 = keep, 0 = ignore
    *pout_item++     = position;
}

static __global__ void decode_kernel_v8_pose_trans(float *predict,
                                                   int    num_bboxes,
                                                   int    pose_num,
                                                   int    output_cdim,
                                                   float  confidence_threshold,
                                                   float *invert_affine_matrix,
                                                   float *parray,
                                                   int    MAX_IMAGE_BOXES,
                                                   int    NUM_BOX_ELEMENT) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes)
        return;

    float *pitem      = predict + output_cdim * position;
    float  confidence = *(pitem + 4);
    int    label      = 0;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= MAX_IMAGE_BOXES)
        return;

    float cx        = *pitem++;
    float cy        = *pitem++;
    float width     = *pitem++;
    float height    = *pitem++;
    float box_score = *pitem++;  // 这句其实没起什么作用，只是简单的让pitem自增
    float left      = cx - width * 0.5f;
    float top       = cy - height * 0.5f;
    float right     = cx + width * 0.5f;
    float bottom    = cy + height * 0.5f;
    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++     = left;
    *pout_item++     = top;
    *pout_item++     = right;
    *pout_item++     = bottom;
    *pout_item++     = confidence;
    *pout_item++     = label;
    *pout_item++     = 1;  // 1 = keep, 0 = ignore
    for (int i = 0; i < pose_num; i++) {
        affine_project(invert_affine_matrix, *pitem++, *pitem++, pout_item++, pout_item++);
        *pout_item++ = *pitem++;  // pose score 赋值
    }
}

static __global__ void decode_single_mask_kernel(int            left,
                                                 int            top,
                                                 float         *mask_weights,
                                                 float         *mask_predict,
                                                 int            mask_width,
                                                 int            mask_height,
                                                 unsigned char *mask_out,
                                                 int            mask_dim,
                                                 int            out_width,
                                                 int            out_height) {
    // mask_predict to mask_out
    // mask_weights @ mask_predict
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx >= out_width || dy >= out_height)
        return;

    int sx = left + dx;
    int sy = top + dy;
    if (sx < 0 || sx >= mask_width || sy < 0 || sy >= mask_height) {
        mask_out[dy * out_width + dx] = 0;
        return;
    }

    float cumprod = 0;
    for (int ic = 0; ic < mask_dim; ++ic) {
        float cval = mask_predict[(ic * mask_height + sy) * mask_width + sx];
        float wval = mask_weights[ic];
        cumprod += cval * wval;
    }

    float alpha = 1.0f / (1.0f + exp(-cumprod));
    // mask_out[dy * out_width + dx] = alpha;
    if (alpha > 0.5)
        mask_out[dy * out_width + dx] = 1;
    else
        mask_out[dy * out_width + dx] = 0;
}

void decode_detect_yolov8_kernel_invoker(float       *predict,
                                         int          num_bboxes,
                                         int          num_classes,
                                         int          output_cdim,
                                         float        confidence_threshold,
                                         float       *invert_affine_matrix,
                                         float       *parray,
                                         int          MAX_IMAGE_BOXES,
                                         hipStream_t stream) {
    auto grid  = CUDATools::grid_dims(num_bboxes);
    auto block = CUDATools::block_dims(num_bboxes);
    checkCudaKernel(decode_kernel_v8_trans<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
}

void decode_pose_yolov8_kernel_invoker(float       *predict,
                                       int          num_bboxes,
                                       int          pose_num,
                                       int          output_cdim,
                                       float        confidence_threshold,
                                       float       *invert_affine_matrix,
                                       float       *parray,
                                       int          MAX_IMAGE_BOXES,
                                       int          NUM_BOX_ELEMENT,
                                       hipStream_t stream) {
    auto grid  = CUDATools::grid_dims(num_bboxes);
    auto block = CUDATools::block_dims(num_bboxes);
    checkCudaKernel(decode_kernel_v8_pose_trans<<<grid, block, 0, stream>>>(
        predict, num_bboxes, pose_num, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES, NUM_BOX_ELEMENT));
}

void decode_single_mask(float          left,
                        float          top,
                        float         *mask_weights,
                        float         *mask_predict,
                        int            mask_width,
                        int            mask_height,
                        unsigned char *mask_out,
                        int            mask_dim,
                        int            out_width,
                        int            out_height,
                        hipStream_t   stream) {
    // mask_weights is mask_dim(32 element) gpu pointer
    dim3 grid((out_width + 31) / 32, (out_height + 31) / 32);
    dim3 block(32, 32);

    checkCudaKernel(decode_single_mask_kernel<<<grid, block, 0, stream>>>(
        left, top, mask_weights, mask_predict, mask_width, mask_height, mask_out, mask_dim,
        out_width, out_height));
}

}  // namespace CUDA