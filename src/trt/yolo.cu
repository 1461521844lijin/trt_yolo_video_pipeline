#include "hip/hip_runtime.h"
#include "infer.hpp"
#include "yolo.hpp"

namespace yolo {

using namespace std;

#define GPU_BLOCK_THREADS 512
#define checkRuntime(call)                                                                 \
  do {                                                                                     \
    auto ___call__ret_code__ = (call);                                                     \
    if (___call__ret_code__ != hipSuccess) {                                              \
      INFO("CUDA Runtime error💥 %s # %s, code = %s [ %d ]", #call,                         \
           hipGetErrorString(___call__ret_code__), hipGetErrorName(___call__ret_code__), \
           ___call__ret_code__);                                                           \
      abort();                                                                             \
    }                                                                                      \
  } while (0)

#define checkKernel(...)                 \
  do {                                   \
    { (__VA_ARGS__); }                   \
    checkRuntime(hipPeekAtLastError()); \
  } while (0)

enum class NormType : int { None = 0, MeanStd = 1, AlphaBeta = 2 };

enum class ChannelType : int { None = 0, SwapRB = 1 };

/* 归一化操作，可以支持均值标准差，alpha beta，和swap RB */
struct Norm {
  float mean[3];
  float std[3];
  float alpha, beta;
  NormType type = NormType::None;
  ChannelType channel_type = ChannelType::None;

  // out = (x * alpha - mean) / std
  static Norm mean_std(const float mean[3], const float std[3], float alpha = 1 / 255.0f,
                       ChannelType channel_type = ChannelType::None);

  // out = x * alpha + beta
  static Norm alpha_beta(float alpha, float beta = 0, ChannelType channel_type = ChannelType::None);

  // None
  static Norm None();
};

Norm Norm::mean_std(const float mean[3], const float std[3], float alpha,
                    ChannelType channel_type) {
  Norm out;
  out.type = NormType::MeanStd;
  out.alpha = alpha;
  out.channel_type = channel_type;
  memcpy(out.mean, mean, sizeof(out.mean));
  memcpy(out.std, std, sizeof(out.std));
  return out;
}

Norm Norm::alpha_beta(float alpha, float beta, ChannelType channel_type) {
  Norm out;
  out.type = NormType::AlphaBeta;
  out.alpha = alpha;
  out.beta = beta;
  out.channel_type = channel_type;
  return out;
}

Norm Norm::None() { return Norm(); }

const int NUM_BOX_ELEMENT = 8;  // left, top, right, bottom, confidence, class,
                                // keepflag, row_index(output)
const int MAX_IMAGE_BOXES = 1024;
inline int upbound(int n, int align = 32) { return (n + align - 1) / align * align; }
static __host__ __device__ void affine_project(float *matrix, float x, float y, float *ox,
                                               float *oy) {
  *ox = matrix[0] * x + matrix[1] * y + matrix[2];
  *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel_common(float *predict, int num_bboxes, int num_classes,
                                            int output_cdim, float confidence_threshold,
                                            float *invert_affine_matrix, float *parray,
                                            int MAX_IMAGE_BOXES) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= num_bboxes) return;

  float *pitem = predict + output_cdim * position;
  float objectness = pitem[4];
  if (objectness < confidence_threshold) return;

  float *class_confidence = pitem + 5;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_classes; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }

  confidence *= objectness;
  if (confidence < confidence_threshold) return;

  int index = atomicAdd(parray, 1);
  if (index >= MAX_IMAGE_BOXES) return;

  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;
  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  affine_project(invert_affine_matrix, left, top, &left, &top);
  affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

  float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;  // 1 = keep, 0 = ignore
}

static __global__ void decode_kernel_v8(float *predict, int num_bboxes, int num_classes,
                                        int output_cdim, float confidence_threshold,
                                        float *invert_affine_matrix, float *parray,
                                        int MAX_IMAGE_BOXES) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= num_bboxes) return;

  float *pitem = predict + output_cdim * position;
  float *class_confidence = pitem + 4;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_classes; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }
  if (confidence < confidence_threshold) return;

  int index = atomicAdd(parray, 1);
  if (index >= MAX_IMAGE_BOXES) return;

  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;
  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  affine_project(invert_affine_matrix, left, top, &left, &top);
  affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

  float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;  // 1 = keep, 0 = ignore
  *pout_item++ = position;
}

static __device__ float box_iou(float aleft, float atop, float aright, float abottom, float bleft,
                                float btop, float bright, float bbottom) {
  float cleft = max(aleft, bleft);
  float ctop = max(atop, btop);
  float cright = min(aright, bright);
  float cbottom = min(abottom, bbottom);

  float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
  if (c_area == 0.0f) return 0.0f;

  float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
  float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
  return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float *bboxes, int MAX_IMAGE_BOXES, float threshold) {
  int position = (blockDim.x * blockIdx.x + threadIdx.x);
  int count = min((int)*bboxes, MAX_IMAGE_BOXES);
  if (position >= count) return;

  // left, top, right, bottom, confidence, class, keepflag
  float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
  for (int i = 0; i < count; ++i) {
    float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
    if (i == position || pcurrent[5] != pitem[5]) continue;

    if (pitem[4] >= pcurrent[4]) {
      if (pitem[4] == pcurrent[4] && i < position) continue;

      float iou = box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0], pitem[1],
                          pitem[2], pitem[3]);

      if (iou > threshold) {
        pcurrent[6] = 0;  // 1=keep, 0=ignore
        return;
      }
    }
  }
}

static dim3 grid_dims(int numJobs) {
  int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
  return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
}

static dim3 block_dims(int numJobs) {
  return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
}

static void decode_kernel_invoker(float *predict, int num_bboxes, int num_classes, int output_cdim,
                                  float confidence_threshold, float nms_threshold,
                                  float *invert_affine_matrix, float *parray, int MAX_IMAGE_BOXES,
                                  Type type, hipStream_t stream) {
  auto grid = grid_dims(num_bboxes);
  auto block = block_dims(num_bboxes);

  if (type == Type::V8 || type == Type::V8Seg) {
    checkKernel(decode_kernel_v8<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
  } else {
    checkKernel(decode_kernel_common<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
  }

  grid = grid_dims(MAX_IMAGE_BOXES);
  block = block_dims(MAX_IMAGE_BOXES);
  checkKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, MAX_IMAGE_BOXES, nms_threshold));
}

static __global__ void warp_affine_bilinear_and_normalize_plane_kernel(
    uint8_t *src, int src_line_size, int src_width, int src_height, float *dst, int dst_width,
    int dst_height, uint8_t const_value_st, float *warp_affine_matrix_2_3, Norm norm) {
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= dst_width || dy >= dst_height) return;

  float m_x1 = warp_affine_matrix_2_3[0];
  float m_y1 = warp_affine_matrix_2_3[1];
  float m_z1 = warp_affine_matrix_2_3[2];
  float m_x2 = warp_affine_matrix_2_3[3];
  float m_y2 = warp_affine_matrix_2_3[4];
  float m_z2 = warp_affine_matrix_2_3[5];

  float src_x = m_x1 * dx + m_y1 * dy + m_z1;
  float src_y = m_x2 * dx + m_y2 * dy + m_z2;
  float c0, c1, c2;

  if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
    // out of range
    c0 = const_value_st;
    c1 = const_value_st;
    c2 = const_value_st;
  } else {
    int y_low = floorf(src_y);
    int x_low = floorf(src_x);
    int y_high = y_low + 1;
    int x_high = x_low + 1;

    uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
    float ly = src_y - y_low;
    float lx = src_x - x_low;
    float hy = 1 - ly;
    float hx = 1 - lx;
    float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    uint8_t *v1 = const_value;
    uint8_t *v2 = const_value;
    uint8_t *v3 = const_value;
    uint8_t *v4 = const_value;
    if (y_low >= 0) {
      if (x_low >= 0) v1 = src + y_low * src_line_size + x_low * 3;

      if (x_high < src_width) v2 = src + y_low * src_line_size + x_high * 3;
    }

    if (y_high < src_height) {
      if (x_low >= 0) v3 = src + y_high * src_line_size + x_low * 3;

      if (x_high < src_width) v4 = src + y_high * src_line_size + x_high * 3;
    }

    // same to opencv
    c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
    c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
    c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
  }

  if (norm.channel_type == ChannelType::SwapRB) {
    float t = c2;
    c2 = c0;
    c0 = t;
  }

  if (norm.type == NormType::MeanStd) {
    c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
    c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
    c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
  } else if (norm.type == NormType::AlphaBeta) {
    c0 = c0 * norm.alpha + norm.beta;
    c1 = c1 * norm.alpha + norm.beta;
    c2 = c2 * norm.alpha + norm.beta;
  }

  int area = dst_width * dst_height;
  float *pdst_c0 = dst + dy * dst_width + dx;
  float *pdst_c1 = pdst_c0 + area;
  float *pdst_c2 = pdst_c1 + area;
  *pdst_c0 = c0;
  *pdst_c1 = c1;
  *pdst_c2 = c2;
}

static void warp_affine_bilinear_and_normalize_plane(uint8_t *src, int src_line_size, int src_width,
                                                     int src_height, float *dst, int dst_width,
                                                     int dst_height, float *matrix_2_3,
                                                     uint8_t const_value, const Norm &norm,
                                                     hipStream_t stream) {
  dim3 grid((dst_width + 31) / 32, (dst_height + 31) / 32);
  dim3 block(32, 32);

  checkKernel(warp_affine_bilinear_and_normalize_plane_kernel<<<grid, block, 0, stream>>>(
      src, src_line_size, src_width, src_height, dst, dst_width, dst_height, const_value,
      matrix_2_3, norm));
}

static __global__ void decode_single_mask_kernel(int left, int top, float *mask_weights,
                                                 float *mask_predict, int mask_width,
                                                 int mask_height, unsigned char *mask_out,
                                                 int mask_dim, int out_width, int out_height) {
  // mask_predict to mask_out
  // mask_weights @ mask_predict
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= out_width || dy >= out_height) return;

  int sx = left + dx;
  int sy = top + dy;
  if (sx < 0 || sx >= mask_width || sy < 0 || sy >= mask_height) {
    mask_out[dy * out_width + dx] = 0;
    return;
  }

  float cumprod = 0;
  for (int ic = 0; ic < mask_dim; ++ic) {
    float cval = mask_predict[(ic * mask_height + sy) * mask_width + sx];
    float wval = mask_weights[ic];
    cumprod += cval * wval;
  }

  float alpha = 1.0f / (1.0f + exp(-cumprod));
  mask_out[dy * out_width + dx] = alpha * 255;
}

static void decode_single_mask(float left, float top, float *mask_weights, float *mask_predict,
                               int mask_width, int mask_height, unsigned char *mask_out,
                               int mask_dim, int out_width, int out_height, hipStream_t stream) {
  // mask_weights is mask_dim(32 element) gpu pointer
  dim3 grid((out_width + 31) / 32, (out_height + 31) / 32);
  dim3 block(32, 32);

  checkKernel(decode_single_mask_kernel<<<grid, block, 0, stream>>>(
      left, top, mask_weights, mask_predict, mask_width, mask_height, mask_out, mask_dim, out_width,
      out_height));
}

const char *type_name(Type type) {
  switch (type) {
    case Type::V5:
      return "YoloV5";
    case Type::V3:
      return "YoloV3";
    case Type::V7:
      return "YoloV7";
    case Type::X:
      return "YoloX";
    case Type::V8:
      return "YoloV8";
    default:
      return "Unknow";
  }
}

struct AffineMatrix {
  float i2d[6];  // image to dst(network), 2x3 matrix
  float d2i[6];  // dst to image, 2x3 matrix

  void compute(const std::tuple<int, int> &from, const std::tuple<int, int> &to) {
    float scale_x = get<0>(to) / (float)get<0>(from);
    float scale_y = get<1>(to) / (float)get<1>(from);
    float scale = std::min(scale_x, scale_y);
    i2d[0] = scale;
    i2d[1] = 0;
    i2d[2] = -scale * get<0>(from) * 0.5 + get<0>(to) * 0.5 + scale * 0.5 - 0.5;
    i2d[3] = 0;
    i2d[4] = scale;
    i2d[5] = -scale * get<1>(from) * 0.5 + get<1>(to) * 0.5 + scale * 0.5 - 0.5;

    double D = i2d[0] * i2d[4] - i2d[1] * i2d[3];
    D = D != 0. ? double(1.) / D : double(0.);
    double A11 = i2d[4] * D, A22 = i2d[0] * D, A12 = -i2d[1] * D, A21 = -i2d[3] * D;
    double b1 = -A11 * i2d[2] - A12 * i2d[5];
    double b2 = -A21 * i2d[2] - A22 * i2d[5];

    d2i[0] = A11;
    d2i[1] = A12;
    d2i[2] = b1;
    d2i[3] = A21;
    d2i[4] = A22;
    d2i[5] = b2;
  }
};

InstanceSegmentMap::InstanceSegmentMap(int width, int height) {
  this->width = width;
  this->height = height;
  checkRuntime(hipHostMalloc(&this->data, width * height));
}

InstanceSegmentMap::~InstanceSegmentMap() {
  if (this->data) {
    checkRuntime(hipHostFree(this->data));
    this->data = nullptr;
  }
  this->width = 0;
  this->height = 0;
}

class InferImpl : public Infer {
 public:
  shared_ptr<trt::Infer> trt_;
  string engine_file_;
  Type type_;
  float confidence_threshold_;
  float nms_threshold_;
  vector<shared_ptr<trt::Memory<unsigned char>>> preprocess_buffers_;
  trt::Memory<float> input_buffer_, bbox_predict_, output_boxarray_;
  trt::Memory<float> segment_predict_;
  int network_input_width_, network_input_height_;
  Norm normalize_;
  vector<int> bbox_head_dims_;
  vector<int> segment_head_dims_;
  int num_classes_ = 0;
  bool has_segment_ = false;
  bool isdynamic_model_ = false;
  vector<shared_ptr<trt::Memory<unsigned char>>> box_segment_cache_;

  virtual ~InferImpl() = default;

  void adjust_memory(int batch_size) {
    // the inference batch_size
    size_t input_numel = network_input_width_ * network_input_height_ * 3;
    input_buffer_.gpu(batch_size * input_numel);
    bbox_predict_.gpu(batch_size * bbox_head_dims_[1] * bbox_head_dims_[2]);
    output_boxarray_.gpu(batch_size * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));
    output_boxarray_.cpu(batch_size * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));

    if (has_segment_)
      segment_predict_.gpu(batch_size * segment_head_dims_[1] * segment_head_dims_[2] *
                           segment_head_dims_[3]);

    if ((int)preprocess_buffers_.size() < batch_size) {
      for (int i = preprocess_buffers_.size(); i < batch_size; ++i)
        preprocess_buffers_.push_back(make_shared<trt::Memory<unsigned char>>());
    }
  }

  void preprocess(int ibatch, const Image &image,
                  shared_ptr<trt::Memory<unsigned char>> preprocess_buffer, AffineMatrix &affine,
                  void *stream = nullptr) {
    affine.compute(make_tuple(image.width, image.height),
                   make_tuple(network_input_width_, network_input_height_));

    size_t input_numel = network_input_width_ * network_input_height_ * 3;
    float *input_device = input_buffer_.gpu() + ibatch * input_numel;
    size_t size_image = image.width * image.height * 3;
    size_t size_matrix = upbound(sizeof(affine.d2i), 32);
    uint8_t *gpu_workspace = preprocess_buffer->gpu(size_matrix + size_image);
    float *affine_matrix_device = (float *)gpu_workspace;
    uint8_t *image_device = gpu_workspace + size_matrix;

    uint8_t *cpu_workspace = preprocess_buffer->cpu(size_matrix + size_image);
    float *affine_matrix_host = (float *)cpu_workspace;
    uint8_t *image_host = cpu_workspace + size_matrix;

    // speed up
    hipStream_t stream_ = (hipStream_t)stream;
    memcpy(image_host, image.bgrptr, size_image);
    memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
    checkRuntime(
        hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream_));
    checkRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(affine.d2i),
                                 hipMemcpyHostToDevice, stream_));

    warp_affine_bilinear_and_normalize_plane(image_device, image.width * 3, image.width,
                                             image.height, input_device, network_input_width_,
                                             network_input_height_, affine_matrix_device, 114,
                                             normalize_, stream_);
  }

  bool load(const string &engine_file, Type type, float confidence_threshold, float nms_threshold) {
    trt_ = trt::load(engine_file);
    if (trt_ == nullptr) return false;

    trt_->print();

    this->type_ = type;
    this->confidence_threshold_ = confidence_threshold;
    this->nms_threshold_ = nms_threshold;

    auto input_dim = trt_->static_dims(0);
    bbox_head_dims_ = trt_->static_dims(1);
    has_segment_ = type == Type::V8Seg;
    if (has_segment_) {
      bbox_head_dims_ = trt_->static_dims(2);
      segment_head_dims_ = trt_->static_dims(1);
    }
    network_input_width_ = input_dim[3];
    network_input_height_ = input_dim[2];
    isdynamic_model_ = trt_->has_dynamic_dim();

    if (type == Type::V5 || type == Type::V3 || type == Type::V7) {
      normalize_ = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
      num_classes_ = bbox_head_dims_[2] - 5;
    } else if (type == Type::V8) {
      normalize_ = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
      num_classes_ = bbox_head_dims_[2] - 4;
    } else if (type == Type::V8Seg) {
      normalize_ = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
      num_classes_ = bbox_head_dims_[2] - 4 - segment_head_dims_[1];
    } else if (type == Type::X) {
      // float mean[] = {0.485, 0.456, 0.406};
      // float std[]  = {0.229, 0.224, 0.225};
      // normalize_ = Norm::mean_std(mean, std, 1/255.0f, ChannelType::SwapRB);
      normalize_ = Norm::None();
      num_classes_ = bbox_head_dims_[2] - 5;
    } else {
      INFO("Unsupport type %d", type);
    }
    return true;
  }

  virtual BoxArray forward(const Image &image, void *stream = nullptr) override {
    auto output = forwards({image}, stream);
    if (output.empty()) return {};
    return output[0];
  }

  virtual vector<BoxArray> forwards(const vector<Image> &images, void *stream = nullptr) override {
    int num_image = images.size();
    if (num_image == 0) return {};

    auto input_dims = trt_->static_dims(0);
    int infer_batch_size = input_dims[0];
    if (infer_batch_size != num_image) {
      if (isdynamic_model_) {
        infer_batch_size = num_image;
        input_dims[0] = num_image;
        if (!trt_->set_run_dims(0, input_dims)) return {};
      } else {
        if (infer_batch_size < num_image) {
          INFO(
              "When using static shape model, number of images[%d] must be "
              "less than or equal to the maximum batch[%d].",
              num_image, infer_batch_size);
          return {};
        }
      }
    }
    adjust_memory(infer_batch_size);

    vector<AffineMatrix> affine_matrixs(num_image);
    hipStream_t stream_ = (hipStream_t)stream;
    for (int i = 0; i < num_image; ++i)
      preprocess(i, images[i], preprocess_buffers_[i], affine_matrixs[i], stream);

    float *bbox_output_device = bbox_predict_.gpu();
    vector<void *> bindings{input_buffer_.gpu(), bbox_output_device};

    if (has_segment_) {
      bindings = {input_buffer_.gpu(), segment_predict_.gpu(), bbox_output_device};
    }

    if (!trt_->forward(bindings, stream)) {
      INFO("Failed to tensorRT forward.");
      return {};
    }

    for (int ib = 0; ib < num_image; ++ib) {
      float *boxarray_device =
          output_boxarray_.gpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
      float *affine_matrix_device = (float *)preprocess_buffers_[ib]->gpu();
      float *image_based_bbox_output =
          bbox_output_device + ib * (bbox_head_dims_[1] * bbox_head_dims_[2]);
      checkRuntime(hipMemsetAsync(boxarray_device, 0, sizeof(int), stream_));
      decode_kernel_invoker(image_based_bbox_output, bbox_head_dims_[1], num_classes_,
                            bbox_head_dims_[2], confidence_threshold_, nms_threshold_,
                            affine_matrix_device, boxarray_device, MAX_IMAGE_BOXES, type_, stream_);
    }
    checkRuntime(hipMemcpyAsync(output_boxarray_.cpu(), output_boxarray_.gpu(),
                                 output_boxarray_.gpu_bytes(), hipMemcpyDeviceToHost, stream_));
    checkRuntime(hipStreamSynchronize(stream_));

    vector<BoxArray> arrout(num_image);
    int imemory = 0;
    for (int ib = 0; ib < num_image; ++ib) {
      float *parray = output_boxarray_.cpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
      int count = min(MAX_IMAGE_BOXES, (int)*parray);
      BoxArray &output = arrout[ib];
      output.reserve(count);
      for (int i = 0; i < count; ++i) {
        float *pbox = parray + 1 + i * NUM_BOX_ELEMENT;
        int label = pbox[5];
        int keepflag = pbox[6];
        if (keepflag == 1) {
          Box result_object_box(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], label);
          if (has_segment_) {
            int row_index = pbox[7];
            int mask_dim = segment_head_dims_[1];
            float *mask_weights = bbox_output_device +
                                  (ib * bbox_head_dims_[1] + row_index) * bbox_head_dims_[2] +
                                  num_classes_ + 4;

            float *mask_head_predict = segment_predict_.gpu();
            float left, top, right, bottom;
            float *i2d = affine_matrixs[ib].i2d;
            affine_project(i2d, pbox[0], pbox[1], &left, &top);
            affine_project(i2d, pbox[2], pbox[3], &right, &bottom);

            float box_width = right - left;
            float box_height = bottom - top;

            float scale_to_predict_x = segment_head_dims_[3] / (float)network_input_width_;
            float scale_to_predict_y = segment_head_dims_[2] / (float)network_input_height_;
            int mask_out_width = box_width * scale_to_predict_x + 0.5f;
            int mask_out_height = box_height * scale_to_predict_y + 0.5f;

            if (mask_out_width > 0 && mask_out_height > 0) {
              if (imemory >= (int)box_segment_cache_.size()) {
                box_segment_cache_.push_back(std::make_shared<trt::Memory<unsigned char>>());
              }

              int bytes_of_mask_out = mask_out_width * mask_out_height;
              auto box_segment_output_memory = box_segment_cache_[imemory];
              result_object_box.seg =
                  make_shared<InstanceSegmentMap>(mask_out_width, mask_out_height);

              unsigned char *mask_out_device = box_segment_output_memory->gpu(bytes_of_mask_out);
              unsigned char *mask_out_host = result_object_box.seg->data;
              decode_single_mask(left * scale_to_predict_x, top * scale_to_predict_y, mask_weights,
                                 mask_head_predict + ib * segment_head_dims_[1] *
                                                         segment_head_dims_[2] *
                                                         segment_head_dims_[3],
                                 segment_head_dims_[3], segment_head_dims_[2], mask_out_device,
                                 mask_dim, mask_out_width, mask_out_height, stream_);
              checkRuntime(hipMemcpyAsync(mask_out_host, mask_out_device,
                                           box_segment_output_memory->gpu_bytes(),
                                           hipMemcpyDeviceToHost, stream_));
            }
          }
          output.emplace_back(result_object_box);
        }
      }
    }

    if (has_segment_) checkRuntime(hipStreamSynchronize(stream_));

    return arrout;
  }
};

Infer *loadraw(const std::string &engine_file, Type type, float confidence_threshold,
               float nms_threshold) {
  InferImpl *impl = new InferImpl();
  if (!impl->load(engine_file, type, confidence_threshold, nms_threshold)) {
    delete impl;
    impl = nullptr;
  }
  return impl;
}

shared_ptr<Infer> load(const string &engine_file, Type type, float confidence_threshold,
                       float nms_threshold) {
  return std::shared_ptr<InferImpl>(
      (InferImpl *)loadraw(engine_file, type, confidence_threshold, nms_threshold));
}

std::tuple<uint8_t, uint8_t, uint8_t> hsv2bgr(float h, float s, float v) {
  const int h_i = static_cast<int>(h * 6);
  const float f = h * 6 - h_i;
  const float p = v * (1 - s);
  const float q = v * (1 - f * s);
  const float t = v * (1 - (1 - f) * s);
  float r, g, b;
  switch (h_i) {
    case 0:
      r = v, g = t, b = p;
      break;
    case 1:
      r = q, g = v, b = p;
      break;
    case 2:
      r = p, g = v, b = t;
      break;
    case 3:
      r = p, g = q, b = v;
      break;
    case 4:
      r = t, g = p, b = v;
      break;
    case 5:
      r = v, g = p, b = q;
      break;
    default:
      r = 1, g = 1, b = 1;
      break;
  }
  return make_tuple(static_cast<uint8_t>(b * 255), static_cast<uint8_t>(g * 255),
                    static_cast<uint8_t>(r * 255));
}

std::tuple<uint8_t, uint8_t, uint8_t> random_color(int id) {
  float h_plane = ((((unsigned int)id << 2) ^ 0x937151) % 100) / 100.0f;
  float s_plane = ((((unsigned int)id << 3) ^ 0x315793) % 100) / 100.0f;
  return hsv2bgr(h_plane, s_plane, 1);
}

};  // namespace yolo